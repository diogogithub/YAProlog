#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
//#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <stdlib.h>
#include "memory.h"

__global__ void marcar(int *dop1, int rows, int cols, int *cons, int numc, int *res) /*a libreria*/
{
 	extern __shared__ int shared[];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int x, rowact, posact;
	if(threadIdx.x < numc)
		shared[threadIdx.x] = cons[threadIdx.x];
	__syncthreads();
	if(id < rows)
	{
		rowact = id * cols;
		for(x = 0; x < numc; x += 2)
		{
			posact = rowact + shared[x];
			if(dop1[posact] != shared[x+1])
				return;
		}
		res[id] = 1;
	}
}

__global__ void marcar2(int *dop1, int rows, int cols, int *cons, int numc, int *res) /*a libreria*/
{
 	extern __shared__ int shared[];
    	int *spos = &shared[numc];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int x, rowact, posact;
	if(threadIdx.x < (numc * 2))
		shared[threadIdx.x] = cons[threadIdx.x];
	__syncthreads();
	if(id < rows)
	{
		if(res[id] == 0)
			return;
		rowact = id * cols;
		for(x = 0; x < numc; x++)
		{
			posact = rowact + spos[x];
			if(dop1[posact] != shared[x])
			{
				res[id] = 0;
				return;
			}
		}
	}
}

__global__ void samejoin(int *dop1, int rows, int cols, int *dhead, int cont, int *res)
{
	extern __shared__ int shared[];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int temp, temp2, pos, x, y;
	if(threadIdx.x < cont)
		shared[threadIdx.x] = dhead[threadIdx.x];
	__syncthreads();
	if(id < rows)
	{	
		if(res[id] == 0)
			return;
		pos = id * cols;
		for(x = 0; x < cont; x++)
		{
			temp = shared[x];
			y = x + 1;
			temp2 = shared[y];
			while(temp2 > -1)
			{
				if(dop1[temp+pos] != dop1[temp2+pos])
				{
					res[id] = 0;
					return;
				}
				y++;
				temp2 = shared[y];
			}
			x = y;
		}
	}
}

__global__ void samejoin2(int *dop1, int rows, int cols, int *dhead, int cont, int *res)
{
	extern __shared__ int shared[];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int temp, temp2, pos, x, y;
	if(threadIdx.x < cont)
		shared[threadIdx.x] = dhead[threadIdx.x];
	__syncthreads();
	if(id < rows)
	{	
		pos = id * cols;
		for(x = 0; x < cont; x++)
		{
			temp = shared[x];
			y = x + 1;
			temp2 = shared[y];
			while(temp2 > -1)
			{
				if(dop1[temp+pos] != dop1[temp2+pos])
					return;
				y++;
				temp2 = shared[y];
			}
			x = y;
		}
		res[id] = 1;
	}
}

__global__ void proyectar(int *dop1, int rows, int cols, int *dhead, int hsize, int *res)
{
	extern __shared__ int shared[];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int pos, posr, x;
	if(threadIdx.x < hsize)
		shared[threadIdx.x] = dhead[threadIdx.x];
	__syncthreads();
	if(id < rows)
	{	
		pos = id * cols;
		posr = id * hsize;
		for(x = 0; x < hsize; x++, posr++)
			res[posr] = dop1[pos+shared[x]];
	}
}

__global__ void llenarproyectar(int *dop1, int rows, int cols, int *temp, int *dhead, int hsize, int *res)
{
	extern __shared__ int shared[];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int pos, posr, x;
	if(threadIdx.x < cols)
		shared[threadIdx.x] = dhead[threadIdx.x];
	__syncthreads();
	if(id < rows)
	{		
		posr = temp[id+1];
		if(temp[id] != posr && posr > 0)
		{
			pos = id * cols;
			posr = (posr - 1) * hsize;			
			for(x = 0; x < hsize; x++, posr++)
				res[posr] = dop1[pos+shared[x]];
		}
	}
}

/*__global__ void removedup()
{
	extern __shared__ int shared[];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(threadIdx.x < cols)
		shared[threadIdx.x] = dhead[threadIdx.x];
	if(id < rows)
	{
		
	}
}*/

template<typename T> /*a libreria*/
struct suma : public binary_function<T,T,T>
{
	__host__ __device__ 
	T operator()(const T &r1, const T &r2)
	{
		if(r1 > -1)
		{
			if(r2 > 0)
				return r1 + r2;
			return -r1;
		}
		else
		{
			if(r2 > 0)
				return abs(r1) + r2;
			return r1;
		}
	}
};

int mayor(int a, int b, int c)
{
	if(a > b)
	{
		if(a > c)
			return a;
	}
	else
	{
		if(b > c)
			return b;
	}
	return c;
}

int selectproyect(int *dop1, int rows, int cols, int head_size, int *select, int numselect, int *selfjoin, int numselfj, int *project, int **ret)
{
	int *fres = NULL, *temp = NULL;
	int *dhead = NULL, tmplen;
	int size, size2, num;
	thrust::device_ptr<int> res;

#if TIMER
	cuda_stats.selects++;
#endif
	int head_bytes = mayor(numselect, numselfj, head_size) * sizeof(int);
	reservar(&dhead, head_bytes);
#ifdef DEBUG_MEM
	cerr << "+ " << dhead << " dhead  " << head_bytes << endl;
#endif

	int blockllen = rows / 1024 + 1;
	int numthreads = 1024;

	//removerep(dop1, rows, cols, dhead,) 
	if(numselect > 0)
	{		
		tmplen = rows + 1;
		size2 = tmplen * sizeof(int);
		reservar(&temp, size2);
#ifdef DEBUG_MEM
		cerr << "+ " << temp << " temp  select " << size2 << endl;
#endif
		hipMemset(temp, 0, size2);

		size = numselect * sizeof(int);
		hipMemcpy(dhead, select, size, hipMemcpyHostToDevice);

		marcar<<<blockllen, numthreads, size>>>(dop1, rows, cols, dhead, numselect, temp + 1);
		
		if(numselfj > 0)
		{
			size = numselfj * sizeof(int);
			hipMemcpy(dhead, selfjoin, size, hipMemcpyHostToDevice);
			samejoin<<<blockllen, numthreads, size>>>(dop1, rows, cols, dhead, numselfj, temp + 1);
		}

		res = thrust::device_pointer_cast(temp);
		thrust::inclusive_scan(res + 1, res + tmplen, res + 1);
		num = res[rows];
		if(num == 0)
			return 0;

		size = head_size * sizeof(int);
		reservar(&fres, num * size);
#ifdef DEBUG_MEM
		cerr << "+ " << fres << " fres select  " << num*size << endl;
#endif
		hipMemcpy(dhead, project, size, hipMemcpyHostToDevice);
		llenarproyectar<<<blockllen, numthreads, size>>>(dop1, rows, cols, temp, dhead, head_size, fres);
		liberar(dhead, head_bytes);
		liberar(temp, size2);
		*ret = fres;
		return num;
	}
	else
	{
		if(numselfj > 0)
		{
			tmplen = rows + 1;
			size2 = tmplen * sizeof(int);
			reservar(&temp, size2);
#ifdef DEBUG_MEM
			cerr << "+ " << temp << " temp select  " << size2 << endl;
#endif
			hipMemset(temp, 0, size2);
			
			size = numselfj * sizeof(int);
			hipMemcpy(dhead, selfjoin, size, hipMemcpyHostToDevice);
			samejoin2<<<blockllen, numthreads, size>>>(dop1, rows, cols, dhead, numselfj, temp + 1);

			res = thrust::device_pointer_cast(temp);
			thrust::inclusive_scan(res + 1, res + tmplen, res + 1);
			num = res[rows];
			if(num == 0)
				return 0;

			size = head_size * sizeof(int);
			reservar(&fres, num * size);
#ifdef DEBUG_MEM
			cerr << "+ " << fres << " fres select again  " << num*size << endl;
#endif
			hipMemcpy(dhead, project, size, hipMemcpyHostToDevice);
			llenarproyectar<<<blockllen, numthreads, size>>>(dop1, rows, cols, temp, dhead, head_size, fres);
			liberar(dhead, head_bytes);
			liberar(temp, size2);
			*ret = fres;
			return num;
		}
		else
		{
			size = head_size * sizeof(int);
			reservar(&fres, rows * size);
#ifdef DEBUG_MEM
			cerr << "+ " << fres << " fres select third  " << rows*size << endl;
#endif
			hipMemcpy(dhead, project, size, hipMemcpyHostToDevice);
			proyectar<<<blockllen, numthreads, size>>>(dop1, rows, cols, dhead, head_size, fres);
			liberar(dhead, head_bytes);
			*ret = fres;
			return rows;
		}
	}
}
