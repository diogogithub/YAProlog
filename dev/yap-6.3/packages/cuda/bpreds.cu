#include "hip/hip_runtime.h"
__global__ void predicates(int *dop1, int rows, int cols, int *cons, int numc, int *res)
{
 	extern __shared__ int shared[];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int x, rowact, op1, op2;
	if(threadIdx.x < numc)
		shared[threadIdx.x] = cons[threadIdx.x];
	__syncthreads();
	if(id < rows)
	{
		rowact = id * cols;
		for(x = 0; x < numc; x += 3)
		{
			op1 = shared[x+1];
			if(op1 < 0)
				op1 *= -1;
			else
				op1 = dop1[rowact + op1];
			op2 = shared[x+2];
			if(op2 < 0)
				op2 *= -1;
			else
				op2 = dop1[rowact + op2];
			switch(shared[x])
			{
				case SBG_EQ:  if(op1 != op2)
						return;
				  break;
				case SBG_GT: if(op1 <= op2)
						return;
				  break;
				case SBG_LT: if(op1 >= op2)
						return;
				  break;
				case SBG_GE: if(op1 < op2)
						return;
				  break;
				case SBG_LE: if(op1 > op2)
						return;
				  break;
				case SBG_DF: if(op1 == op2)
						return;
			}
		}
		res[id] = 1;
	}
}

int bpreds(int *dop1, int rows, int cols, int *bin, int3 numpreds, int **ret)
{
	int *temp;
	int tmplen = rows + 1;
	int size = tmplen * sizeof(int);
	reservar(&temp, size);
#ifdef DEBUG_MEM
	 cerr << "+ " << temp << " temp bpreds " << size << endl;
#endif
	hipMemset(temp, 0, size);

#if TIMER
	cuda_stats.builtins++;
#endif
	int *dhead;
	int predn = numpreds.x * 3;
	int spredn = predn * sizeof(int);
	int sproj = numpreds.z * sizeof(int);
	int hsize;
	if(predn > numpreds.z)
		hsize = spredn;
	else
		hsize = sproj;
	reservar(&dhead, hsize);
#ifdef DEBUG_MEM
	cerr << "+ " << dhead << " dhead  " << hsize << endl;
#endif
	hipMemcpy(dhead, bin, spredn, hipMemcpyHostToDevice);

	int blockllen = rows / 1024 + 1;
	int numthreads = 1024;

	/*int x;
	cout << "arraypreds = ";
	for(x = 0; x < predn; x++)
		cout << bin[x] << " ";
	cout << endl;
	cout << "temptable = ";
	for(x = 0; x < numpreds.z; x++)
		cout << bin[x+predn] << " ";
	cout << endl; 
	int y;
	int *hop1 = (int *)malloc(numpreds.y * rows * sizeof(int));
	hipMemcpy(hop1, dop1, numpreds.y * rows * sizeof(int), hipMemcpyDeviceToHost);
	for(x = 0; x < rows; x++)
	{
		for(y = 0; y < numpreds.y; y++)
			cout << hop1[x * numpreds.y + y] << " ";
		cout << endl;
	}
	free(hop1);*/

	predicates<<<blockllen, numthreads, spredn>>>(dop1, rows, numpreds.y, dhead, predn, temp + 1);

	/*int y;
	int *hop1 = (int *)malloc((rows + 1) * sizeof(int));
	hipMemcpy(hop1, temp, (rows + 1) * sizeof(int), hipMemcpyDeviceToHost);
	for(x = 0; x < (rows + 1); x++)
		cout << hop1[x] << " ";
	cout << endl;
	free(hop1);*/

	thrust::device_ptr<int> res;
	res = thrust::device_pointer_cast(temp);
	thrust::inclusive_scan(res + 1, res + tmplen, res + 1);
	int num = res[rows];
	if(num == 0)
		return 0;

	int *fres;
	reservar(&fres, num * sproj);
#ifdef DEBUG_MEM
	cerr << "+ " << fres << " fres  " << num * sproj << endl;
#endif
	hipMemcpy(dhead, bin + predn, sproj, hipMemcpyHostToDevice);
	llenarproyectar<<<blockllen, numthreads, sproj>>>(dop1, rows, numpreds.y, temp, dhead, numpreds.z, fres);

	/*int y;
	int *hop1 = (int *)malloc(numpreds.z * num * sizeof(int));
	hipMemcpy(hop1, fres, numpreds.z * num * sizeof(int), hipMemcpyDeviceToHost);
	for(x = 0; x < num; x++)
	{
		for(y = 0; y < numpreds.z; y++)
			cout << hop1[x * numpreds.z + y] << " ";
		cout << endl;
	}
	free(hop1);*/

	liberar(dhead, hsize);
	liberar(temp, size);
	liberar(dop1, rows * cols * sizeof(int));

	*ret = fres;
	return num;
}
