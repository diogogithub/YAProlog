#include <list>
#include <iostream>
#include <stdlib.h>
#include <algorithm>
#include <thrust/device_vector.h>
#include "lista.h"
#include "memory.h"

#define MAX_REC 200
#define HALF_REC (MAX_REC / 2)
#define MAX_FIX_POINTS 100

unsigned int avmem;
memnode temp_storage[MAX_REC];
list<memnode> GPUmem;
list<memnode> CPUmem;

bool compareiteration(const memnode &r1, const memnode &r2)
{
	return (r1.iteration < r2.iteration); 
}

bool comparename(const memnode &r1, const memnode &r2)
{
	return (r1.name > r2.name); 
}

void calcular_mem(int dev)
{
	hipDeviceProp_t p;
	hipGetDeviceProperties(&p, dev);
	avmem = p.totalGlobalMem;
	temp_storage[0].dev_address = NULL;
	temp_storage[0].size = 0;
	temp_storage[HALF_REC].dev_address = NULL;
	temp_storage[HALF_REC].size = 0;

	//cout << "Initial memory available " << avmem << endl;
}

template<class InputIterator>
InputIterator buscarhecho(InputIterator first, InputIterator last, int name)
{
	while(first!=last) 
	{
		if(first->name == name) return first;
			++first;
	}
	return last;
}

list<memnode>::iterator buscarpornombre(int name, int itr, int *totalrows, int *gpunum)
{
	int x = 1, sum = 0;
	memnode temp;

	temp.name = name;
	temp.iteration = itr;
	pair<list<memnode>::iterator, list<memnode>::iterator> rec = equal_range(GPUmem.begin(), GPUmem.end(), temp, compareiteration);
	while(rec.first != rec.second)
	{		
		
		//cout << "itr = " << itr << " rec.first = " << rec.first->name << endl;	
		
		if(rec.first->name == name)
		{
			temp_storage[x] = *rec.first;
			rec.first = GPUmem.erase(rec.first);
			sum += temp_storage[x].rows;
			x++;
		}	
		else
			rec.first++;
	}
	//if(x > 1)
	rec.first = GPUmem.insert(rec.first, temp);
	*totalrows = sum;
	*gpunum = x;
	return rec.first;
}

int buscarpornombrecpu(int name, int itr, int *totalrows)
{
	int x = HALF_REC + 1, sum = 0;
	memnode temp;
	temp.iteration = itr;
	pair<list<memnode>::iterator, list<memnode>::iterator> rec = equal_range(CPUmem.begin(), CPUmem.end(), temp, compareiteration);

	/*if(rec.first != rec.second)
		cout << "bscnomcpu = " << rec.first->name << " " << rec.first->iteration << endl;*/

	while(rec.first != rec.second)
	{
		if(rec.first->name == name)
		{
			temp_storage[x] = *rec.first;
			rec.first = CPUmem.erase(rec.first);
			sum += temp_storage[x].rows;
			x++;
		}	
		else
			rec.first++;
	}
	*totalrows += sum;
	return x;
}

void limpiar(const char s[], size_t sz)
{
	list<memnode>::iterator ini;
	memnode temp;

	if(GPUmem.size() == 0)
	{
		cerr << s << ": not enough GPU memory: have " << avmem << ", need " << sz << " bytes." << endl;
		exit(1);
	}		

	ini = GPUmem.begin();
	if(ini->isrule)
	{	
		temp = *ini;
		temp.dev_address = (int *)malloc(ini->size);
		hipMemcpyAsync(temp.dev_address, ini->dev_address, temp.size, hipMemcpyDeviceToHost);
		CPUmem.push_back(temp);
	}
	liberar(ini->dev_address, ini->size);
	GPUmem.erase(ini);
}

void limpiartodo(int *p1, int *p2)
{
	list<memnode>::iterator ini;
	memnode temp;
	int cont = 0;
	if(p1 != NULL)
		cont++;	
	if(p2 != NULL)
		cont++;
	ini = GPUmem.begin();

	/*cout << "ANTES" << endl;
	mostrar_memoria();
	mostrar_memcpu();
	cout << "FIN ANTES" << endl;*/
	//cout << "mem = " << GPUmem.size() << " " << avmem << endl;

	while(GPUmem.size() > cont)
	{
		if(ini->dev_address == p1 || ini->dev_address == p2)
		{
			ini++;
			continue;
		}
		if(ini->isrule)
		{
			temp = *ini; 
			temp.dev_address = (int *)malloc(ini->size);
			hipMemcpy(temp.dev_address, ini->dev_address, temp.size, hipMemcpyDeviceToHost);
			CPUmem.push_back(temp);
		}
		liberar(ini->dev_address, temp.size);
		ini = GPUmem.erase(ini);
	}

	/*cout << "DESPUES" << endl;
	mostrar_memoria();
	mostrar_memcpu();
	cout << "FIN DESPUES" << endl;*/
	//cout << "memfinal = " << GPUmem.size() << " " << avmem << endl;

}

void liberar(int *ptr, int size)
{
	//cout << "L " << avmem << " " << size; 

	hipFree(ptr);
#ifdef DEBUG_MEM
	cerr << "- " << ptr << " " << size << endl;
#endif
	avmem += size;
	
	//cout << " " << avmem << endl;
}

void reservar(int **ptr, int size)
{
  //size_t free, total;
  //hipMemGetInfo(      &free, &total	 );
  //	cerr << "? " << free << " " << size << endl;

        if (size == 0) { 
                *ptr = NULL; 
                return;
        }
	while(avmem < size)
		limpiar("not enough memory", size);
	while(hipMalloc(ptr, size) == hipErrorOutOfMemory)
		limpiar("Error in memory allocation", size);
	if (! *ptr ) {
	  size_t free, total;
	  hipMemGetInfo(      &free, &total	 );
	  cerr << "Could not allocate " << size << " bytes, only " << free << " avaliable from total of " << total << " !!!" << endl;
	  cerr << "Exiting CUDA...." << endl;
	  exit(1);
	}
	avmem -= size;

	// cout << " " << avmem << endl;
}

void registrar(int name, int num_columns, int *ptr, int rows, int itr, int rule)
{
	memnode temp;
	temp.name = name;
	temp.dev_address = ptr;
	temp.rows = rows;
	temp.size = rows * num_columns * sizeof(int);
	temp.iteration = itr;
	temp.isrule = rule;
	GPUmem.push_back(temp);
}

template<class InputIterator>
void actualizar(int num_columns, int *ptr, int rows, InputIterator i)
{
	i->dev_address = ptr;
	i->rows = rows;
	i->size = rows * num_columns * sizeof(int);
}

int numrows(int name, int itr)
{
	int sum = 0;
	memnode temp;
	temp.iteration = itr;
	pair<list<memnode>::iterator, list<memnode>::iterator> rec = equal_range(GPUmem.begin(), GPUmem.end(), temp, compareiteration);
	while(rec.first != rec.second)
	{
		if(rec.first->name == name)
			sum += rec.first->rows;
		rec.first++;
	}
	rec = equal_range(CPUmem.begin(), CPUmem.end(), temp, compareiteration);
	while(rec.first != rec.second)
	{
		if(rec.first->name == name)
			sum += rec.first->rows;
		rec.first++;
	}
	return sum;
}


	extern "C" void * YAP_IntToAtom(int);
	extern  "C" char * YAP_AtomName(void *);


int cargar(int name, int num_rows, int num_columns, int is_fact, int *address_host_table, int **ptr, int itr)
{
	int numgpu, numcpu, totalrows = 0;
	int *temp, x;
	int size, itrant;
	list<memnode>::iterator i;
	memnode fact;

	if(is_fact)
	{
		i = buscarhecho(GPUmem.begin(), GPUmem.end(), name);
		if(i != GPUmem.end())
		{
			fact = *i;
			GPUmem.erase(i);
			fact.iteration = itr;
			*ptr = fact.dev_address;
			GPUmem.push_back(fact);
			return fact.rows;
		}
		size = num_rows * num_columns * sizeof(int);
		reservar(&temp, size);
#ifdef DEBUG_MEM
		cerr << "+ " << temp << " temp  " << size << endl;
#endif
		hipMemcpyAsync(temp, address_host_table, size, hipMemcpyHostToDevice);
		registrar(name, num_columns, temp, num_rows, itr, 0);
		*ptr = temp;
		return num_rows;
	}
	if(itr > 0)
	{
		itrant = itr - 1;
		i = buscarpornombre(name, itrant, &totalrows, &numgpu);
		numcpu = buscarpornombrecpu(name, itrant, &totalrows);

		if((numgpu == 2) && (numcpu == (HALF_REC + 1)))
		{
			actualizar(num_columns, temp_storage[1].dev_address, temp_storage[1].rows, i);
			*ptr = temp_storage[1].dev_address;
			return temp_storage[1].rows;
		}
		size = totalrows * num_columns * sizeof(int);
		reservar(&temp, size);
#ifdef DEBUG_MEM
		cerr << "+ " << temp << " temp 2  " << size << endl;
#endif
		for(x = 1; x < numgpu; x++)
		{
			hipMemcpyAsync(temp + temp_storage[x-1].size, temp_storage[x].dev_address, temp_storage[x].size, hipMemcpyDeviceToDevice);
			liberar(temp_storage[x].dev_address, temp_storage[x].size);
		}
		for(x = HALF_REC + 1; x < numcpu; x++)
		{
			hipMemcpyAsync(temp + temp_storage[x-1].size, temp_storage[x].dev_address, temp_storage[x].size, hipMemcpyHostToDevice);
			free(temp_storage[x].dev_address);
		}
		actualizar(num_columns, temp, totalrows, i);
		*ptr = temp;
		return totalrows;
	}
	return 0;
}

int cargafinal(int name, int cols, int **ptr)
{
	int *temp, *ini, cont = 0;
	memnode bus;
	bus.name = name;
	GPUmem.sort(comparename);
	CPUmem.sort(comparename);
	list<memnode>::iterator endg = GPUmem.end();
	list<memnode>::iterator endc = CPUmem.end();
	list<memnode>::iterator pos = lower_bound(GPUmem.begin(), endg, bus, comparename);
	list<memnode>::iterator gpu = pos;
	while(pos != endg && pos->name == name)
	{
		cont += pos->rows;
		pos++;
	}
	pos = lower_bound(CPUmem.begin(), endc, bus, comparename);
	list<memnode>::iterator cpu = pos;
	while(pos != endc && pos->name == name)
	{
		cont += pos->rows;
		pos++;
	}
	
	reservar(&temp, cont * cols * sizeof(int));
#ifdef DEBUG_MEM
	cerr << "+ " << temp << " temp 3 " << cont * cols * sizeof(int) << endl;
#endif
	ini = temp;	

	pos = gpu;
	while(pos != endg && pos->name == name)
	{
		hipMemcpy(temp, pos->dev_address, pos->size, hipMemcpyDeviceToDevice);
		temp += pos->size / sizeof(int);
		pos++;
	}
	pos = cpu;
	while(pos != endc && pos->name == name)
	{
		hipMemcpy(temp, pos->dev_address, pos->size, hipMemcpyHostToDevice);
		temp += pos->size / sizeof(int);
		pos++;
	}

	/*int x, y;
	int *hop1 = (int *)malloc(cont * cols * sizeof(int));
	hipMemcpy(hop1, ini, cont * cols * sizeof(int), hipMemcpyDeviceToHost);
	cout << "select finala" << endl;
	for(x = 0; x < cont; x++)
	{
		for(y = 0; y < cols; y++)
			cout << hop1[x * cols + y] << " ";
		cout << endl;
	}
	cout << "select finala" << endl;*/

	*ptr = ini;
	return cont;
}

bool generadas(int name, int filas, int cols, int itr)
{
	int r1, r2, x, fin;
	int *dop1, *dop2;

	r2 = numrows(name, itr);
	if(itr < MAX_FIX_POINTS)
		fin = itr;
	else
		fin = MAX_FIX_POINTS;
	for(x = 1; x <= fin; x++)
	{
		r1 = numrows(name, itr - x);
		if(r1 == r2)
		{
			r2 = cargar(name, filas, cols, 0, NULL, &dop2, itr + 1);
			thrust::device_ptr<int> pt2 = thrust::device_pointer_cast(dop2);
			r1 = cargar(name, filas, cols, 0, NULL, &dop1, itr - x + 1);
			thrust::device_ptr<int> pt1 = thrust::device_pointer_cast(dop1);

			/*int y;
			int *a = (int *)malloc(r1 * cols * sizeof(int));
			hipMemcpy(a, dop1, r1 * cols * sizeof(int), hipMemcpyDeviceToHost);
			for(x = 0; x < r1; x++)
			{
				for(y = 0; y < cols; y++)
					cout << a[x * cols + y] << " ";
			}
			cout << endl;
			hipMemcpy(a, dop2, r1 * cols * sizeof(int), hipMemcpyDeviceToHost);
			for(x = 0; x < r1; x++)
			{
				for(y = 0; y < cols; y++)
					cout << a[x * cols + y] << " ";
			}
			cout << endl;
			free(a);*/

			if(thrust::equal(pt1, pt1 + r1, pt2) == true)
				return true;
		}
	}

	return false;
}

void mostrar_memoria()
{
	int x;
	list<memnode>::iterator i = GPUmem.begin();
	cout << "Memoria inicio GPU" << endl;
	for(x = 0; x < GPUmem.size(); x++, i++)
		cout << i->name << " " << i->iteration << " " << i->size << endl;
	cout << "Memoria fin GPU" << endl;
}

void mostrar_memcpu()
{
	int x;
	list<memnode>::iterator i = CPUmem.begin();
	cout << "Memoria inicio CPU" << endl;
	for(x = 0; x < CPUmem.size(); x++, i++)
		cout << i->name << " " << i->iteration << endl;
	cout << "Memoria fin CPU" << endl;
}

void resultados(vector<rulenode>::iterator first, vector<rulenode>::iterator last)
{
	GPUmem.sort(comparename);
	CPUmem.sort(comparename);
	list<memnode>::iterator gpu = GPUmem.begin();
	list<memnode>::iterator cpu = CPUmem.begin();
	int x, y, of, cols;
	int *temp, cont = 0;
	while(first != last)
	{
		while(first->name == gpu->name)
		{
			temp = (int *)malloc(gpu->size);
			hipMemcpy(temp, gpu->dev_address, gpu->size, hipMemcpyDeviceToHost);
			cols = gpu->size / (gpu->rows * sizeof(int));
			cont += gpu->rows;
			for(x = 0, of = 0; x < gpu->rows; x++)
			{
				for(y = 0; y < cols; y++, of++)
					cout << temp[of] << " ";
				cout << endl;
			}
			hipFree(gpu->dev_address);
#ifdef DEBUG_MEM
			cerr << "- " << gpu->dev_address << " gpu->dev_address" << endl;
#endif
			free(temp);
			gpu++;
		}
		while(first->name == cpu->name)
		{
			cols = cpu->size / (cpu->rows * sizeof(int));
			cont += cpu->rows;
			for(x = 0, of = 0; x < cpu->rows; x++)
			{
				for(y = 0; y < cols; y++, of++)
					cout << cpu->dev_address[of] << " ";
				cout << endl;
			}
			free(cpu->dev_address);
			cpu++;
		}
		first++;
	}
	cout << cont << endl;
}

void clear_memory()
{
	list<memnode>::iterator ini;
	list<memnode>::iterator fin;
       	ini = GPUmem.begin();
	fin = GPUmem.end();
	while(ini != fin)
	{
	  if (ini->isrule) {
	    hipFree(ini->dev_address);
#ifdef DEBUG_MEM
	    cerr << "- " << ini->dev_address << " ini->dev_address" << endl;
#endif
	    ini = GPUmem.erase(ini);
	  } else {
	    ini++;
	  }
	}
	ini = CPUmem.begin();
	fin = CPUmem.end();
	while(ini != fin)
	{
		free(ini->dev_address);
		ini++;
	}
	CPUmem.clear();
}
