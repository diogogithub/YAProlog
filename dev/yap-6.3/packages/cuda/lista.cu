#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include "lista.h"
#include "memory.h"
extern "C" {
#include "pred.h"
}
#include "selectproyect.cu"
#include "treeb.cu"
#include "union2.cu"
#include "bpreds.cu"

#define MAXVALS 200

#if TIMER
statinfo cuda_stats;
#endif

bool compare(const gpunode &r1, const gpunode &r2)
{
	return (r1.name > r2.name); 
}

bool comparecompleted(const compnode &r1, const compnode &r2)
{
	return (r1.name > r2.name); 
}

bool comparef(const gpunode &r1, const gpunode &r2)
{
	return (r1.is_fact > r2.is_fact); 
}

bool comparer(const rulenode &r1, const rulenode &r2)
{
	return (r1.name > r2.name); 
}

template<class InputIterator>
InputIterator buscar(InputIterator inicio, InputIterator rul_str, InputIterator fin, int name)
{
	vector<gpunode>::iterator i;
	gpunode temp;
	temp.name = name;
	i = lower_bound(inicio, rul_str, temp, compare);
	if (i != rul_str && i->name == name)
        	return i;
	i = lower_bound(rul_str, fin, temp, compare);
	if (i != rul_str && i->name == name)
        	return i;
	return fin;
}

void buscarreglas(vector<gpunode> *facts, vector<rulenode> *rules)
{
	vector<gpunode>::iterator first = facts->begin();
	rulenode temp;
	while(first != facts->end())
	{
		if(!first->is_fact)
		{
			temp.name = first->name;
			temp.num_rows = first->num_rows;
			temp.address_host_table = first->address_host_table;
			rules->push_back(temp);
			first = facts->erase(first);
		}
		else
			first++;
	}
}

template<class InputIterator>
void movebpreds(InputIterator rules, InputIterator end)
{
	int x, subs, total, cont, cont2, pos;
	int *move, *rest;
	while(rules != end)
	{
		if(rules->num_bpreds.x > 0)
		{

			total = rules->num_rows+rules->num_bpreds.x;

			/*cout << "ANTES" << endl;
			for(x = 0; x < rules->rule_names[total]; x++)
				cout << rules->address_host_table[x] << " ";
			cout << "FINANTES" << endl;*/

			move = (int *)malloc(sizeof(int) * rules->num_bpreds.x * 4);
			rest = (int *)malloc(sizeof(int) * rules->rule_names[total]);
			cont = 0;
			cont2 = 0;
			for(x = 0; x < total; x++)
			{
				subs = rules->rule_names[x+1] - rules->rule_names[x];
				
				//cout << subs << " ";
				
				if(rules->address_host_table[rules->rule_names[x]] > 0)
				{
					memcpy(rest + cont, rules->address_host_table + rules->rule_names[x], subs * sizeof(int));
					cont += subs;
				}
				else
				{
					memcpy(move + cont2, rules->address_host_table + rules->rule_names[x], subs * sizeof(int));
					cont2 += subs;
				}
			}

			/*cout << "REST" << endl;
			for(x = 0; x < cont; x++)
				cout << rest[x] << " ";
			cout << "RESTFIN" << endl;*/

			memcpy(rest + cont, move, cont2 * sizeof(int));
			pos = 1;
			for(x = 1; x <= total; x++)
			{
				while(rest[pos] != 0)
					pos++;
				pos++;
				rules->rule_names[x] = pos;
			}
			memcpy(rules->address_host_table, rest, sizeof(int) * rules->rule_names[total]);
			free(move);
			free(rest);

			/*cout << "DESPUES" << endl;
			for(x = 0; x < rules->rule_names[total]; x++)
				cout << rules->address_host_table[x] << " ";
			cout << "FINDESPUES" << endl;*/

		}
		rules++;
	}
}

template<class InputIterator>
void nombres(InputIterator rules, InputIterator end)
{
	int x, pos;
	while(rules != end)
	{
		pos = 1;
		rules->rule_names = (int *)malloc(sizeof(int) * (rules->num_rows + 1));
		rules->rule_names[0] = 0;
		rules->num_bpreds.x = 0;
		for(x = 1; x <= rules->num_rows; x++)
		{
			while(rules->address_host_table[pos] != 0)
				pos++;
			pos++;
			rules->rule_names[x] = pos;
			if(rules->address_host_table[pos] < 0 && x < rules->num_rows)
				rules->num_bpreds.x++;
		}
		rules->num_columns = rules->rule_names[1] - 2;
		rules->num_rows -= rules->num_bpreds.x; /*quita los builtin del total*/
		rules++;
	}
}

template<class InputIterator, class RulesIterator>
void referencias(InputIterator facts, InputIterator fend, RulesIterator rules, RulesIterator end)
{
	vector<gpunode>::iterator res1;
	gpunode n1;
	vector<rulenode>::iterator actual = rules, res2;
	rulenode n2;
	int x, temp, cont;
	while(actual != end)
	{
		actual->referencias = (int *)malloc((actual->num_rows - 1) * sizeof(int));
		cont = 0;
		for(x = 1; x < actual->num_rows; x++)
		{
			temp = actual->address_host_table[actual->rule_names[x]];
			if(temp < 0) /*puede ser reemplazado una vez que se reacomoden las reglas*/
				continue;
			if(temp == actual->name)
				actual->referencias[x-1] = actual - rules;
			else
			{
				n1.name = temp;
				res1 = lower_bound(facts, fend, n1, compare);
				if (res1 != fend && res1->name == temp)
				{
					actual->referencias[x-1] = -(res1 - facts) - 1;
					cont++;
				}
				else
				{
					n2.name = temp;
					res2 = lower_bound(rules, end, n2, comparer);
					actual->referencias[x-1] = res2 - rules;
				}
			}
		}
		if(cont == (x - 1))
			actual->gen_ant = 0;
		else
			actual->gen_ant = -1;
		actual->gen_act = 0;
		actual++;
	}
}

template<class InputIterator>
void seleccion(InputIterator actual, InputIterator end)
{
	int x, y, ini, temp, sl, tam, cont;
	int pv[MAXVALS];
	while(actual != end)
	{
		temp = actual->num_rows - 1;
		actual->select = (int **)malloc(temp * sizeof(int *));
		actual->numsel = (int *)malloc(temp * sizeof(int));
		for(x = 1; x < actual->num_rows; x++)
		{
			ini = actual->rule_names[x];
			if(actual->address_host_table[ini] < 0)
				continue;
			ini++;
			cont = 0;
			for(y = ini; y < (actual->rule_names[x+1] - 1); y++)
			{
				temp = actual->address_host_table[y];
				if(temp < 0)
				{
					pv[cont] = y - ini;
					cont++;
					pv[cont] = -temp;
					cont++;
				}
			}
			sl = x - 1;
			tam = cont * sizeof(int);
			actual->select[sl] = (int *)malloc(tam);
			memcpy(actual->select[sl], pv, tam);
			actual->numsel[sl] = cont;
		}
		actual++;
	}
}

int notin(int bus, int *array, int size)
{
	int x;
	for(x = 0; x < size; x++)
	{
		if(array[x] == bus)
			return 0;
	}
	return 1;
}

int2 columnsproject(int *first, int tam, int *rule, int ini, int fin, int sini, int sfin, int **res, int **newrule)
{
	int x, y, temp;
	int pv[MAXVALS], pv2[MAXVALS];
	int2 ret = make_int2(0, 0);
	for(x = 0; x < tam; x++)
	{
		y = 0;
		temp = first[x];
		for(y = 0; y < ini; y++)
		{
			if(temp == rule[y] && temp > 0) /*added condition to avoid constants*/
			{
				if(notin(temp, pv, ret.y))
				{
					pv[ret.y] = temp;
					pv2[ret.y] = x;
					ret.y++;
				}
				break;
			}
		}
		if(y != ini)
			continue;
		for(y = sfin + 1; y < fin; y++)
		{
			if(temp == rule[y] && temp > 0)
			{
				if(notin(temp, pv, ret.y))
				{
					pv[ret.y] = temp;
					pv2[ret.y] = x;
					ret.y++;
				}
				break;
			}
		}
	}
	ret.x = ret.y;
	for(x = sini; x < sfin; x++)
	{
		y = 0;
		temp = rule[x];
		for(y = 0; y < ini; y++)
		{
			if(temp == rule[y] && temp > 0)
			{
				if(notin(temp, pv, ret.y))
				{
					pv[ret.y] = temp;
					pv2[ret.y] = x - sini;
					ret.y++;
				}
				break;
			}
		}
		if(y != ini)
			continue;
		for(y = sfin + 1; y < fin; y++)
		{
			if(temp == rule[y] && temp > 0)
			{
				if(notin(temp, pv, ret.y))
				{
					pv[ret.y] = temp;
					pv2[ret.y] = x - sini;
					ret.y++;
				}
				break;
			}
		}
	}
	temp = ret.y * sizeof(int);
	free(*newrule);
	*newrule = (int *)malloc(temp);
	memcpy(*newrule, pv, temp);
	*res = (int *)malloc(temp);
	memcpy(*res, pv2, temp);	
	return ret;
}

int wherejoin(int *tmprule, int tmplen, int *rule, int tam2, int **res)
{
	int x, y, temp;
	int cont = 0;
	int joins[MAXVALS];
	for(x = 0; x < tmplen; x++)
	{
		for(y = 0; y < tam2; y++)
		{
			if(rule[y] > 0 && tmprule[x] == rule[y])
			{
				joins[cont] = x;
				cont++;
				joins[cont] = y;
				cont++;
				break;
			}
		}
	}

	temp = cont * sizeof(int);
	*res = (int *)malloc(temp);
	memcpy(*res, joins, temp);
	return cont;
}

int builtinpredicates(int *tmprule, int tmplen, int *rule, int ini, int fin, int **res)
{
	int x, y, temp;
	int cont = 0, cont2 = 0;
	int joins[MAXVALS];
	
	for(x = ini; x < fin; x++)
	{
		joins[cont] = rule[x];
		cont++;
		x++;
		if(rule[x] < 0)
		{
			joins[cont] = rule[x];
			cont++;
			x++;
		}
		for(y = 0; y < tmplen; y++)
		{
			if(tmprule[y] == rule[x])
			{
				joins[cont] = y;
				cont++;
				x++;
				break;
			}
		}
		if(rule[x] == 0)
			continue;
		if(rule[x] < 0)
		{
			joins[cont] = rule[x];
			cont++;
			x++;
			continue;
		}
		for(y = 0; y < tmplen; y++)
		{
			if(tmprule[y] == rule[x])
			{
				joins[cont] = y;
				cont++;
				x++;
				break;
			}
		}
	}
	x = 1;
	while(rule[x] != 0)
	{
		for(y = 0; y < tmplen; y++)
		{
			if(rule[x] == tmprule[y])
			{
				joins[cont] = y;
				cont++;
				cont2++;
				break;
			}
		}
		x++;
	}
	temp = cont * sizeof(int);
	*res = (int *)malloc(temp);
	memcpy(*res, joins, temp);
	return cont2;
}

template<class InputIterator>
void proyeccion(InputIterator actual, InputIterator end)
{
	int x, y, ini, fin, total, numjoins, temp, rulestart, ruleend, malptr;
	int fjoin[MAXVALS];
	int *pv, *res;
	int2 pos;
	while(actual != end)
	{
		numjoins = actual->num_rows - 2;
		if(numjoins < 1)
		{
			actual->projpos = (int2 *)malloc(sizeof(int2));
			ini = actual->rule_names[1] + 1;
			fin = actual->rule_names[2] - 1;

			if(actual->numsel[0] == 0 && actual->numselfj[0] == 0 && actual->num_columns == (fin - ini))
			{
				for(x = 1, y = actual->num_columns + 3; x <= actual->num_columns; x++, y++)
				{
					if(actual->address_host_table[x] != actual->address_host_table[y])
						break;
				}
				if(x > actual->num_columns)
				{
					pos.x = -1;
					pos.y = -1;
					actual->projpos[0] = pos;
					actual++;
					continue;
				}
			}
			actual->project = (int **)malloc(sizeof(int *));
			pos.x = 0;
			for(x = 1; x <= actual->num_columns; x++)
			{
				temp = actual->address_host_table[x];
				for(y = ini; y < fin; y++)
				{
					if(temp == actual->address_host_table[y])
					{
						fjoin[pos.x] = y - ini;
						pos.x++;
						break;
					}
				}
			}
			temp = pos.x * sizeof(int);
			actual->project[0] = (int *)malloc(temp);
			memcpy(actual->project[0], fjoin, temp);
			pos.y = pos.x;
			actual->projpos[0] = pos;
			actual++;
			continue;
		}
		malptr = numjoins * sizeof(int *);
		actual->project = (int **)malloc(malptr);
		actual->projpos = (int2 *)malloc(numjoins * sizeof(int2));
		actual->wherejoin = (int **)malloc(malptr);
		actual->numjoin = (int *)malloc(numjoins * sizeof(int));
		ini = actual->rule_names[1] + 1;
		total = actual->num_rows + actual->num_bpreds.x;
		fin = actual->rule_names[total] - 1;
		pos.y = actual->rule_names[2] - actual->rule_names[1] - 2;
		temp = pos.y * sizeof(int);
		pv = (int *)malloc(temp);
		memcpy(pv, actual->address_host_table + actual->rule_names[1] + 1, temp);

		for(x = 2, y = 0; x <= numjoins; x++, y++)
		{
			rulestart = actual->rule_names[x] + 1;
			ruleend = actual->rule_names[x+1] - 1;
			temp = wherejoin(pv, pos.y, actual->address_host_table + rulestart, ruleend - rulestart, &res);
			actual->wherejoin[y] = res;
			actual->numjoin[y] = temp;
			pos = columnsproject(pv, pos.y, actual->address_host_table, ini, fin, rulestart, ruleend, &res, &pv);
			actual->project[y] = res;
			actual->projpos[y] = pos;
		}

		rulestart = actual->rule_names[actual->num_rows-1] + 1;
		ruleend = actual->rule_names[actual->num_rows] - 1; 

		temp = wherejoin(pv, pos.y, actual->address_host_table + rulestart, ruleend - rulestart, &res);
		actual->wherejoin[y] = res;
		actual->numjoin[y] = temp;
		numjoins--;
		
		if(actual->num_bpreds.x > 0)
		{
			pos = columnsproject(pv, pos.y, actual->address_host_table, ini, fin, rulestart, ruleend, &res, &pv);
			actual->project[numjoins] = res;
			actual->projpos[numjoins] = pos;
			actual->num_bpreds.y = pos.y; /*para guardar el tamanio de la union final*/
			actual->num_bpreds.z = builtinpredicates(pv, pos.y, actual->address_host_table, ruleend + 1, actual->rule_names[total] - 1, &res);
			actual->builtin = res;
		}
		else
		{
			pos.x = 0;
			for(x = 1; x <= actual->num_columns; x++)
			{
				temp = actual->address_host_table[x];
				for(y = 0; y < pos.y; y++)
				{
					if(temp == pv[y])
					{
						fjoin[pos.x] = y + 1;
						pos.x++;
						break;
					}
				}
				if(y != pos.y)
					continue;
				for(y = rulestart; y < ruleend; y++)
				{
					if(temp == actual->address_host_table[y])
					{
						fjoin[pos.x] = -(y - rulestart + 1);
						pos.x++;
						break;
					}
				}
			}

			temp = pos.x * sizeof(int);	
			actual->project[numjoins] = (int *)malloc(temp);
			memcpy(actual->project[numjoins], fjoin, temp);
			pos.y = pos.x;
			actual->projpos[numjoins] = pos;
		}
		actual++;
	}
}

template<class InputIterator>
void selfjoin(InputIterator actual, InputIterator end)
{
	int x, y, z;
	int cont, tam, temp, size, pos;
	int fjoin[MAXVALS], rulecpy[MAXVALS];
	while(actual != end)
	{
		size = actual->num_rows - 1;
		actual->selfjoin = (int **)malloc(size * sizeof(int *));
		actual->numselfj = (int *)malloc(size * sizeof(int));
		for(x = 1; x <= size; x++)
		{
			pos = actual->rule_names[x];
			if(actual->address_host_table[pos] < 0)
				continue;
			tam = actual->rule_names[x+1] - actual->rule_names[x] - 2;
			memcpy(rulecpy, actual->address_host_table + pos + 1, tam * sizeof(int));
			cont = 0;
			for(y = 0; y < tam; y++)
			{
				temp = rulecpy[y];
				if(temp > -1)
				{
					for(z = y + 1; z < tam; z++)
					{
						if(temp == rulecpy[z])
						{
							fjoin[cont] = y;
							cont++;
							fjoin[cont] = z;
							cont++;
							rulecpy[z] = -1;
							for(z++; z < tam; z++)
							{
								if(temp == rulecpy[z])
								{
									fjoin[cont] = z;
									cont++;
									rulecpy[z] = -1;
								}
							}
							fjoin[cont] = -1;
							cont++;
						}
					}
				}
			}
			temp = x - 1;
			tam = cont * sizeof(int);
			actual->selfjoin[temp] = (int *)malloc(tam);
			memcpy(actual->selfjoin[temp], fjoin, tam);
			actual->numselfj[temp] = cont;
		}
		actual++;
	}
}

template<class InputIterator>
int linears(InputIterator first, InputIterator last, int name)
{
	while(first != last) 
	{
    		if(first->name == name) 
			return 0;
    		first++;
  	}
  	return 1;
}

template<class InputIterator>
void tempointer(InputIterator rules, InputIterator end, vector<rulenode>::iterator aux)
{
	while(rules != end)
	{
		rules->temp = &(*aux);
		rules++;
		aux++;
	}
}

void cargareglas(vector<rulenode> *rules, int name, list<rulenode> *res) /*This is the function to create the rule queue based on the query*/
{
	rulenode searched;
	vector<rulenode>::iterator ini = rules->begin(), fin = rules->end(), actual;
	list<rulenode>::iterator start;
	unsigned int numrules;
	int x, pos;
	searched.name = name;
	actual = lower_bound(ini, fin, searched, comparer);
	if(actual == fin)
		return;
	while(actual != fin && actual->name == name)
	{
		res->push_back(*actual);
		actual++;
	}
	numrules = rules->size();
	start = res->begin();
	while(res->size() < numrules && start != res->end())
	{
		for(x = 0; x < start->num_rows - 1; x++)
		{
			pos = start->referencias[x];
			if(pos > -1)
			{
				searched = rules->at(pos);
				if(linears(res->begin(), res->end(), searched.name))
					res->push_back(searched);
			}
		}
		start++;
	}
	res->sort(comparer);
}

void consulta(int *query, int qsize, int qname, rulenode *res)
{
	int sel[MAXVALS], pro[MAXVALS];
	int temp, cont1 = 0, cont2 = 0, size;
	int x, y;
	res->numsel = (int *)malloc(sizeof(int));
	res->numselfj = (int *)malloc(sizeof(int));
	for(x = 0; x < qsize; x++)
	{
		temp = query[x];
		if(temp < 0)
		{
			sel[cont1] = x;
			cont1++;
			sel[cont1] = -temp;
			cont1++;
		}
		else
		{
			pro[cont2] = x;
			cont2++;
		}
	}
	res->numsel[0] = cont1;
	res->num_columns = cont2;
	if(cont1 > 0)
	{
		size = cont1 * sizeof(int);
		res->select = (int **)malloc(sizeof(int *));
		res->select[0] = (int *)malloc(size);
		memcpy(res->select[0], sel, size);
		cont1 = 0;
	}
	if(cont2 > 0)
	{
		size = cont2 * sizeof(int);
		res->project = (int **)malloc(sizeof(int *));
		res->project[0] = (int *)malloc(size);
		memcpy(res->project[0], pro, size);
	}
	for(x = 0; x < qsize; x++)
	{
		temp = query[x];
		if(temp > -1)
		{
			for(y = x + 1; y < qsize; y++)
			{
				if(temp == query[y])
				{
					sel[cont1] = x;
					cont1++;
					sel[cont1] = y;
					cont1++;
					query[y] = -1;
					for(y++; y < qsize; y++)
					{
						if(temp == query[y])
						{
							sel[cont1] = y;
							cont1++;
							query[y] = -1;
						}
					}
					sel[cont1] = -1;
					cont1++;
				}
			}
		}
	}
	res->numselfj[0] = cont1;
	if(cont1 > 0)
	{
		size = cont1 * sizeof(int);
		res->selfjoin = (int **)malloc(sizeof(int *));
		res->selfjoin[0] = (int *)malloc(size);
		memcpy(res->selfjoin[0], sel, size);
	}
}

template<class InputIterator>
void completitud(InputIterator actual, InputIterator end, vector<compnode> *rulcomp)
{
	vector<compnode>::iterator bus;
	compnode searched;
	while(actual != end)
	{
		searched.name = actual->name;
		bus = lower_bound(rulcomp->begin(), rulcomp->end(), searched, comparecompleted);
		if(bus == rulcomp->end() || bus->name != searched.name)
		{
			searched.numrules = 1;
			searched.reduce = 0;
			rulcomp->push_back(searched);
		}
		else
			bus->numrules++;
		actual++;
	}
	bus = rulcomp->begin();
	while(bus != rulcomp->end())
	{
		bus->reset = bus->numrules;
		bus++;
	}
}

template<class InputIterator>
void mostrarcontenido(InputIterator actual, InputIterator end)
{
	int y, z, num;
	cout << "AUX INICIO" << endl;
	cout << "tamanio = " << end - actual << endl;
	while(actual != end)
	{
		cout << "name = " << actual->name << endl;
		cout << "rule_names = ";
		for(y = 0; y <= actual->num_rows; y++)
			cout << actual->rule_names[y] << " ";
		cout << endl << "referencias = ";
		num = actual->num_rows - 1;
		for(y = 0; y < num; y++)
			cout << actual->referencias[y] << " ";
		cout << endl << "select = " << endl;
		for(y = 0; y < num; y++)
		{	
			cout << actual->numsel[y] << ": ";
			for(z = 0; z < actual->numsel[y]; z++)
				cout << actual->select[y][z] << " ";
			cout << endl;
		}
		cout << "selfjoins = " << endl;
		for(y = 0; y < num; y++)
		{	
			cout << actual->numselfj[y] << ": ";
			for(z = 0; z < actual->numselfj[y]; z++)
				cout << actual->selfjoin[y][z] << " ";
			cout << endl;
		}
		cout << "project = " << endl;
		for(y = 0; y < num; y++)
		{	
			cout << actual->projpos[y].x << " " << actual->projpos[y].y << ": ";
			for(z = 0; z < actual->projpos[y].y; z++)
				cout << actual->project[y][z] << " ";
			cout << endl;
		}
		num--;
		cout << "wherejoin = " << endl;
		for(y = 0; y < num; y++)
		{	
			cout << actual->numjoin[y] << ": ";
			for(z = 0; z < actual->numjoin[y]; z++)
				cout << actual->wherejoin[y][z] << " ";
			cout << endl;
		}
		actual++;
	}	
	cout << "AUX FIN" << endl;
}

void mostrareglas(list<rulenode> aux)
{
	list<rulenode>::iterator actual = aux.begin();
	cout << "Rules to eval = ";
	while(actual != aux.end())
	{
		cout << actual->name << " ";
		actual++;
	}
	cout << endl;
}

extern "C"
 void Cuda_Statistics(void)
{
  cerr << "GPU Statistics" << endl;
#if TIMER
  cerr << "Called " << cuda_stats.calls << "times." << endl;
  cerr << "GPU time " << cuda_stats.total_time << "msec." << endl;
  cerr << "Longest call " << cuda_stats.max_time << "msec." << endl;
  cerr << "Fastest call " << cuda_stats.min_time << "msec." << endl << endl;
  cerr << "Steps" << endl;
  cerr << "    Select First: " << cuda_stats.select1_time << " msec." << endl;
  cerr << "    Select Second: " << cuda_stats.select2_time << " msec." << endl;
  cerr << "    Sort: " << cuda_stats.sort_time << " msec." << endl;
  cerr << "    Join: " << cuda_stats.join_time << " msec." << endl;
  cerr << "    Union: " << cuda_stats.union_time << " msec." << endl;
  cerr << "    Built-in: " << cuda_stats.pred_time << " msec." << endl << endl;
  cerr << "Operations" << endl;
  cerr << "    Joins: " << cuda_stats.joins << "." << endl;
  cerr << "    Selects/Projects: " << cuda_stats.selects << "." << endl;
  cerr << "    Unions: " << cuda_stats.unions << "." << endl;
  cerr << "    Built-ins: " << cuda_stats.builtins << "." << endl << endl;
#endif
}

extern "C"
int Cuda_Eval(predicate **inpfacts, int ninpf, predicate **inprules, int ninpr, predicate *inpquery, int **result)
{
	vector<gpunode> L;
	int showr = 0; /*1 show results; 0 don't show results*/
	int x, y;
	int qsize, *query, qname;

#if TIMER
	cuda_stats.calls++;
#endif
	for(x = 0; x < ninpf; x++)
		L.push_back(*inpfacts[x]);
	for(x = 0; x < ninpr; x++)
		L.push_back(*inprules[x]);

	/*cout << "NAMES" << endl;
	for(x = 0; x < (ninpf+ninpr); x++)
		cout << L[x].name << endl;
	cout << "NAMESEND" << endl;*/

	qname = inpquery->name;
	query = inpquery->address_host_table;
	qsize = inpquery->num_columns;

	/*cout << qname << " " << qsize << endl;
	for(x = 0; x < q->symbols_num; x++)
		cout << q->symbols_id[x] << " ";
	cout << endl;*/

	vector<gpunode>::iterator i;
	i = L.begin();

	calcular_mem(0);
	int res_rows, rows1, rows2;
	int tipo;
	int *dop1, *dop2, *res;
	
	vector<rulenode> rules;
	vector<rulenode>::iterator rul_str, fin;
	buscarreglas(&L, &rules);
	sort(L.begin(), L.end(), compare);
	sort(rules.begin(), rules.end(), comparer);
	rul_str = rules.begin();
	fin = rules.end();

	nombres(rul_str, fin); /*preprocessing*/
	//movebpreds(rul_str, fin);
	referencias(L.begin(), L.end(), rul_str, fin);
	seleccion(rul_str, fin);
	selfjoin(rul_str, fin);
	proyeccion(rul_str, fin);
	
	//mostrarcontenido(rul_str, fin);

	list<rulenode> reglas;
	list<rulenode>::iterator rul_act, busqueda;
	rulenode completed;
	cargareglas(&rules, qname, &reglas);
	//mostrareglas(reglas);

	gpunode tmpfact;
	rulenode tmprule;
	int name1, filas1, cols1, isfact1, name2, filas2, cols2, isfact2;
	int *table1, *table2, *hres;
	int num_refs, itr = 0;
	vector<gpunode>::iterator qposf;
	vector<rulenode>::iterator qposr;

#if TIMER
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif

	while(reglas.size()) /*Here's the main loop*/
	{
		rul_act = reglas.begin();

		while(rul_act != reglas.end()) /*Here's the loop that evaluates each rule*/
		{
			tipo = rul_act->referencias[0];
			if(tipo)
			{
				tmpfact = L.at(-tipo - 1);
				name1 = tmpfact.name;
				filas1 = tmpfact.num_rows;
				cols1 = tmpfact.num_columns;
				isfact1 = 1;
				table1 = tmpfact.address_host_table;
			}
			else
			{
				tmprule = rules.at(tipo);
				name1 = tmprule.name;
				filas1 = tmprule.num_rows;
				cols1 = tmprule.num_columns;
				isfact1 = 0;
				table1 = NULL;
			}

			rows1 = cargar(name1, filas1, cols1, isfact1, table1, &dop1, itr);

			// cout << "rows1 = " << rows1  << endl;

			if(rows1 == 0)
			{
				//rul_act->gen_ant = rul_act->gen_act;
				rul_act->gen_act = 0;
				rul_act++;
				continue;
			}
			
			if(rul_act->num_rows < 3)
			{	
				if(rul_act->projpos[0].x == -1)
				{
					num_refs = rows1 * cols1 * sizeof(int);
					reservar(&res, num_refs);
#ifdef DEBUG_MEM
					cerr << "+ " << res << " Res  " << num_refs << endl;
#endif
					hipMemcpyAsync(res, dop1, num_refs, hipMemcpyDeviceToDevice);
					registrar(rul_act->name, cols1, res, rows1, itr, 1);
					rul_act->gen_ant = rul_act->gen_act;
					rul_act->gen_act = rows1;
				}
				else
				{

					/*int x, y;
					int *hop1 = (int *)malloc(cols1 * rows1 * sizeof(int));
					hipMemcpy(hop1, dop1, cols1 * rows1 * sizeof(int), hipMemcpyDeviceToHost);
					for(x = 0; x < rows1; x++)
					{
						for(y = 0; y < cols1; y++)
							cout << hop1[x * cols1 + y] << " ";
						cout << endl;
					}
					free(hop1);*/

					res_rows = selectproyect(dop1, rows1, cols1, rul_act->num_columns, rul_act->select[0], rul_act->numsel[0], rul_act->selfjoin[0], rul_act->numselfj[0], rul_act->project[0], &res);
					if(res_rows > 0)
					{
						registrar(rul_act->name, rul_act->num_columns, res, res_rows, itr, 1);
						rul_act->gen_ant = rul_act->gen_act;
						rul_act->gen_act = res_rows;
					}
					else
					{
						//rul_act->gen_ant = rul_act->gen_act;
						rul_act->gen_act = 0;
					}
				}
				rul_act++;
				continue;
			}

			tipo = rul_act->referencias[1];
			if(tipo < 0)
			{
				tmpfact = L.at(-tipo - 1);
				name2 = tmpfact.name;
				filas2 = tmpfact.num_rows;
				cols2 = tmpfact.num_columns;
				isfact2 = 1;
				table2 = tmpfact.address_host_table;
			}
			else
			{
				tmprule = rules.at(tipo);
				name2 = tmprule.name;
				filas2 = tmprule.num_rows;
				cols2 = tmprule.num_columns;
				isfact2 = 0;
				table2 = NULL;
			}

			rows2 = cargar(name2, filas2, cols2, isfact2, table2, &dop2, itr);
			
			//cout << "rows2 = " << rows2 << endl;
	
			if(rows2 == 0)
			{
				//rul_act->gen_ant = rul_act->gen_act;
				rul_act->gen_act = 0;
				rul_act++;
				continue;
			}

			res = NULL;
			res_rows = join(dop1, dop2, rows1, rows2, cols1, cols2, rul_act, 0, 1, &res);
	
			if(res_rows == 0)
			{
				//rul_act->gen_ant = rul_act->gen_act;
				rul_act->gen_act = 0;
				rul_act++;
				continue;
			}

			num_refs = rul_act->num_rows - 1;
			for(x = 2; x < num_refs; x++)
			{
			  if (rul_act->address_host_table[x] < 0) {
					#ifdef TIMER
					hipEvent_t start3, stop3;
					hipEventCreate(&start3);
					hipEventCreate(&stop3);
					hipEventRecord(start3, 0);
					#endif					
				
					res_rows = bpreds(res, res_rows, rul_act->projpos[x-2].y, rul_act->builtin, rul_act->num_bpreds, &res);

					#ifdef TIMER
					hipEventRecord(stop3, 0);
					hipEventSynchronize(stop3);
					hipEventElapsedTime(&time, start3, stop3);
					hipEventDestroy(start3);
					hipEventDestroy(stop3);
					//cout << "Predicados = " << time << endl;
					cuda_stats.pred_time += time;
					#endif
			    continue;
			  }
				tipo = rul_act->referencias[x];
				if(tipo < 0)
				{
					tmpfact = L.at(-tipo - 1);
					name2 = tmpfact.name;
					filas2 = tmpfact.num_rows;
					cols2 = tmpfact.num_columns;
					isfact2 = 1;
					table2 = tmpfact.address_host_table;
				}
				else
				{
					tmprule = rules.at(tipo);
					name2 = tmprule.name;
					filas2 = tmprule.num_rows;
					cols2 = tmprule.num_columns;
					isfact2 = 0;
					table2 = NULL;
				}

				rows2 = cargar(name2, filas2, cols2, isfact2, table2, &dop2, itr);

				//out << "rows = " << x << " " << rows2 << endl;

				if(rows2 == 0)
					break;
				cout << x << ": join = " << res_rows << "/" <<  rul_act->projpos[x-2].y << " " << rows2 << "/" << cols2 << endl;
				res_rows = join(res, dop2, res_rows, rows2, rul_act->projpos[x-2].y, cols2, rul_act, x-1, 0, &res);
				if(res_rows == 0)
					break;
				
				cout << x << ": resrows before = " << res_rows << " cols = " <<  rul_act->projpos[x-1].y << endl;
				if (x < num_refs-1 && res_rows > 32) {
				  
#ifdef TIMER
				  hipEvent_t start2, stop2;
				  hipEventCreate(&start2);
				  hipEventCreate(&stop2);
				  hipEventRecord(start2, 0);
#endif

				  res_rows = unir(res, res_rows, rul_act->projpos[x-1].y); /*Duplicate Elimination*/
#ifdef TIMER
				  hipEventRecord(stop2, 0);
				  hipEventSynchronize(stop2);
				  hipEventElapsedTime(&time, start2, stop2);
				  hipEventDestroy(start2);
				  hipEventDestroy(stop2);
				  //cout << "Union = " << time << endl;
				  cuda_stats.union_time += time;
#endif					
	
				  cout << "resrows after = " << res_rows << endl;
				}

			}

			if(x == num_refs)
			{
				//cout << "antes de unir = " << res_rows << endl;

				#ifdef TIMER
				hipEvent_t start2, stop2;
				hipEventCreate(&start2);
				hipEventCreate(&stop2);
				hipEventRecord(start2, 0);
				#endif

				res_rows = unir(res, res_rows, rul_act->num_columns); /*Duplicate Elimination*/

				#ifdef TIMER
				hipEventRecord(stop2, 0);
				hipEventSynchronize(stop2);
				hipEventElapsedTime(&time, start2, stop2);
				hipEventDestroy(start2);
				hipEventDestroy(stop2);
				//cout << "Union = " << time << endl;
				cuda_stats.union_time += time;
				#endif					
	
				//cout << "despues de unir = " << res_rows << endl;

				registrar(rul_act->name, rul_act->num_columns, res, res_rows, itr, 1);	
				rul_act->gen_ant = rul_act->gen_act;
				rul_act->gen_act = res_rows;
			}
			else
			{
				//rul_act->gen_ant = rul_act->gen_act;
				rul_act->gen_act = 0;
			}
			rul_act++;
		}

		rul_act = reglas.begin();

		/*while(rul_act != reglas.end())
		{
			cout << rul_act->gen_act << " " << rul_act->gen_ant << endl;
			rul_act++;
		}
		return 0;*/

		//cout << rul_act->gen_act << " " << rul_act->gen_ant << endl;

		while(rul_act != reglas.end()) /*Here's the loop that discards finished rules*/
		{
			if(rul_act->gen_act == -1 || rul_act->gen_ant == -1) //&& rul_act->gen_act == 0))
			{
				rul_act++;
				continue;
			}
			if(rul_act->gen_act == 0)
			{
				rul_act->gen_act = -1;
				rul_act++;
				continue;
			}
			num_refs = rul_act->num_rows - 1;
			for(x = 0; x < num_refs; x++)
			{
				tipo = rul_act->referencias[x];
				if(tipo < 0)
					continue;
				completed.name = rul_act->address_host_table[rul_act->rule_names[x+1]];
				if(!binary_search(reglas.begin(), reglas.end(), completed, comparer))
				{
					rul_act->gen_act = -1;
					break;
				}

				//cout << rul_act->gen_act << " " << rul_act->gen_ant << endl;			

				/*if(rul_act->gen_act == rul_act->gen_ant)
				{			
					tipo = rul_act->name;
					busqueda = rul_act;
					busqueda++;
					while(busqueda != reglas.end() && busqueda->name == tipo)
					{
						if(busqueda->gen_act != busqueda->gen_ant)
							break;
						busqueda++;
					}
					if(busqueda != reglas.end() && busqueda->name == tipo)
						break;*/
				tipo = rul_act->name;
				if(generadas(tipo, rul_act->num_rows, rul_act->num_columns, itr))
				{
					rul_act->gen_act = -1;
					busqueda = rul_act;
					busqueda++;
					while(busqueda != reglas.end() && busqueda->name == tipo)
					{
						busqueda->gen_act = -1;
						busqueda++;
					}
				}
				break;
			}
			if(x == num_refs)
				rul_act->gen_act = -1;
			rul_act++;
		}
		rul_act = reglas.begin();
		while(rul_act != reglas.end())
		{
			if(rul_act->gen_act == -1)
				rul_act = reglas.erase(rul_act);
			else
				rul_act++;
		}
		
		//cout << "ITR = " << itr << endl;

		itr++;
	}

	tmprule.name = qname;
	qposr = lower_bound(rul_str, fin, tmprule, comparer);
	if(qposr != fin && qposr->name == qname) 
	{
		cols1 = qposr->num_columns;
		rows1 = cargafinal(qname, cols1, &dop1);
	}
	else
	{
		tmpfact.name = qname;
		qposf = lower_bound(L.begin(), L.end(), tmpfact, compare);
		cols1 = qposf->num_columns;
		rows1 =  cargar(qname, qposf->num_rows, cols1, 1, qposf->address_host_table, &dop1, 0);
	}

	if(rows1 > 0) /*Query consideration*/
	{
		consulta(query + 1, qsize, qname, &tmprule);
		if(tmprule.numsel[0] == 0 && tmprule.numselfj[0] == 0)
		{
			res = dop1;
			res_rows = rows1;
		}
		else
		{		
			res_rows = selectproyect(dop1, rows1, cols1, tmprule.num_columns, tmprule.select[0], tmprule.numsel[0], tmprule.selfjoin[0], tmprule.numselfj[0], tmprule.project[0], &res);
			if(qposr != fin && qposr->name == qname) {
				hipFree(dop1);
#ifdef DEBUG_MEM
				cerr << "- " << dop1 << " dop1" << endl;
#endif
			}
		}

		cols1 = tmprule.num_columns;
		tipo = res_rows * cols1 * sizeof(int);
		hres = (int *)malloc(tipo);
		hipMemcpy(hres, res, tipo, hipMemcpyDeviceToHost);
		if(res_rows > 0 /*&& tmprule.numsel[0] != 0 && tmprule.numselfj[0] != 0 */) {
			hipFree(res);
#ifdef DEBUG_MEM
			cerr << "- " << res << " res" << endl;
#endif
		}
	}
	else
		res_rows = 0;

#if TIMER
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	cuda_stats.total_time += time;
	if (time > cuda_stats.max_time) 
	  cuda_stats.max_time = time;
	if (time < cuda_stats.min_time || cuda_stats.calls == 1) 
	  cuda_stats.min_time = time;
	hipEventDestroy(start);
	hipEventDestroy(stop);
#endif

	if(showr == 1)
	{
		for(x = 0; x < res_rows; x++)
		{
			cols2 = x * cols1 + cols1;
			for(y = x * cols1; y < cols2; y++)
				cout << hres[y] << " ";
			cout << endl;
		}
	}
	//free(hres);

	//cout << "Elapsed = " << time << endl;
	//cout << "Size = " << res_rows << endl;
	//cout << "Iterations = " << itr << endl;

	clear_memory();
	*result = hres;

	return res_rows;
}

	/*gpunode k;
	k.name = 666;
	L.push_back(k);
	k.name = 777;
	L.push_back(k);
	int a = 666;
	vector<gpunode>::iterator i;
   	for(i=L.begin(); i != L.end(); ++i)
		cout << i->name << " ";
	i = buscar(L.begin(), L.end(), a);
	cout << endl << i->name;
	int *pred = (int *)malloc(sizeof(int) * 4);
	pred[0] = 2;
	pred[3] = 6;
	Cuda_newPred(888, 2, 2, pred, &L);
	for(i=L.begin(); i != L.end(); ++i)
		cout << i->name << " ";*/
