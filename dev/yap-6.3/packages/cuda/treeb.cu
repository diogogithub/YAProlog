#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/gather.h>
#include <thrust/scan.h>
#include <iostream>
#include <cstdarg>
#include <cstdio>
#include "lista.h"
//#include "scanImpl.cu"

#define WARP_SIZE		32
#define TREE_NODE_SIZE		WARP_SIZE
#define TREE_FANOUT		(TREE_NODE_SIZE + 1)

#define N_MULTI_P 		16
#define BLCK_PER_MP_create	256	// blocks per multiprocessor during tree creation
#define BLCK_PER_MP_search	512	// blocks per multiprocessor during tree searching
#define WAPRS_PER_BLCK_join	8//16	// blocks per multiprocessor during tree creation
#define BLCK_PER_MP_join	512//256	// blocks per multiprocessor during tree searching

#define THRD_PER_BLCK_create	TREE_NODE_SIZE
#define BLCK_PER_GRID_create	(N_MULTI_P * BLCK_PER_MP_create)
#define THRD_PER_BLCK_search	TREE_NODE_SIZE
#define BLCK_PER_GRID_search	(N_MULTI_P * BLCK_PER_MP_search)
#define THRD_PER_GRID_search	(THRD_PER_BLCK_search * BLCK_PER_GRID_search)
#define THRD_PER_BLCK_join	(WARP_SIZE * WAPRS_PER_BLCK_join)
#define BLCK_PER_GRID_join	(N_MULTI_P * BLCK_PER_MP_join)
#define THRD_PER_GRID_join	(THRD_PER_BLCK_join * BLCK_PER_GRID_join)

#define TEST_MAX		100

typedef int IKeyType;
typedef int Record;

typedef struct {
	int keys[TREE_NODE_SIZE];
} IDirectoryNode;

typedef struct {
	Record records[TREE_NODE_SIZE];
} IDataNode;

typedef struct {
	IDataNode* data;
	unsigned int nDataNodes;
	IDirectoryNode* dir;
	unsigned int nDirNodes;
} CUDA_CSSTree;

__host__ __device__ unsigned int uintCeilingLog(unsigned int base, unsigned int num)
{
	unsigned int result = 0;

	for(unsigned int temp = 1; temp < num; temp *= base)
		result++;

	return result;
}

__host__ __device__ unsigned int uintCeilingDiv(unsigned int dividend, unsigned int divisor)
{
	return (dividend + divisor - 1) / divisor;
}

__host__ __device__ unsigned int uintPower(unsigned int base, unsigned int pow)
{
	unsigned int result = 1;

	for(; pow; pow--)
		result *= base;

	return result;
}

__device__ int getRightMostDescIdx(int tree_size, int nodeIdx)
{
	int tmp = nodeIdx * TREE_NODE_SIZE + TREE_FANOUT;
	int n = uintCeilingLog(TREE_FANOUT, uintCeilingDiv(TREE_NODE_SIZE * tree_size + TREE_FANOUT, tmp)) - 1;

	int result = (tmp * uintPower(TREE_FANOUT, n) - TREE_FANOUT) / TREE_NODE_SIZE;
    	return result; 
}

__device__ int getDataArrayIdx(int dirSize, int tree_size, int bottom_start, int treeIdx)
{
	int idx;
	if(treeIdx < dirSize) {
		idx = tree_size - bottom_start - 1;
	}
	else if( treeIdx < bottom_start ) {
		idx = tree_size - bottom_start + treeIdx - dirSize;
	}
	else {
		idx = treeIdx - bottom_start;
	}
	return idx;
}

// Binary Search
__device__ int firstMatchingKeyInDirNode1(int keys[], int key)
{
	int min = 0;
	int max = TREE_NODE_SIZE;
	int mid;
	int cut;
	while(max - min > 1) {
		mid = (min + max) / 2;
		cut = keys[mid];

		if(key > cut)
			min = mid;
		else
			max = mid;
	}

	if(keys[min] >= key)
		return min;

	return max;

}

// Binary Search
__device__ int firstMatchingKeyInDataNode2(Record records[], IKeyType key)
{
	int min = 0;
	int max = TREE_NODE_SIZE;
	int mid;
	int cut;
	while(max - min > 1) {
		mid = (min + max) / 2;
		cut = records[mid];

		if(key > cut)
			min = mid;
		else
			max = mid;
	}

	if(records[min] == key)
		return min;

	if(max < TREE_NODE_SIZE && records[max] == key)
		return max;

	return -1;
}

__global__ void gCreateIndex(IDataNode data[], IDirectoryNode dir[], int dirSize, int tree_size, int bottom_start, int nNodesPerBlock)
{
        int startIdx = blockIdx.x * nNodesPerBlock;
        int endIdx = startIdx + nNodesPerBlock;
        if(endIdx > dirSize)
                endIdx = dirSize;
        int keyIdx = threadIdx.x;

        // Proceed only when in internal nodes
        for(int nodeIdx = startIdx; nodeIdx < endIdx; nodeIdx++)
        {
                int childIdx = nodeIdx * TREE_FANOUT + keyIdx + 1;        // One step down to the left
                // Then look for the right most descendent
                int rightMostDesIdx;
                // Common cases
                if(childIdx < tree_size) {
                        rightMostDesIdx = getRightMostDescIdx(tree_size, childIdx);
                }
                // versus the unusual case when the tree is incomplete and the node does not have the full set of children
                else {
                        // pick the last node in the tree (largest element of the array)
                        rightMostDesIdx = tree_size - 1;
                }

                int dataArrayIdx = getDataArrayIdx(dirSize, tree_size, bottom_start, rightMostDesIdx);
	        dir[nodeIdx].keys[keyIdx] = data[dataArrayIdx].records[TREE_NODE_SIZE - 1];
        }
}

__global__ void gSearchTree(IDataNode* data, int nDataNodes, IDirectoryNode* dir, int nDirNodes, int lvlDir, Record* arr, int locations[], int nSearchKeys, int nKeysPerThread, int tree_size, int bottom_start)
{
	// Bringing the root node (visited by every tuple) to the faster shared memory
	__shared__ IKeyType RootNodeKeys[TREE_NODE_SIZE];
	RootNodeKeys[threadIdx.x] = dir->keys[threadIdx.x];

	__syncthreads();

	int OverallThreadIdx = blockIdx.x * THRD_PER_BLCK_search + threadIdx.x;

	for(int keyIdx = OverallThreadIdx; keyIdx < nSearchKeys; keyIdx += THRD_PER_GRID_search)
	{
		IKeyType val = arr[keyIdx];
		int loc = firstMatchingKeyInDirNode1(RootNodeKeys, val) + 1;
		for(int i = 1; i < lvlDir && loc < nDirNodes; i++) {
			int kid = firstMatchingKeyInDirNode1(dir[loc].keys, val);
			loc = loc * TREE_FANOUT + kid + 1;
		}

		if(loc >= tree_size)
			loc = nDataNodes - 1;
		else
			loc = getDataArrayIdx(nDirNodes, tree_size, bottom_start, loc);

		int offset = firstMatchingKeyInDataNode2(data[loc].records, val);
		locations[keyIdx] = (offset <0)?-1:(loc * TREE_NODE_SIZE + offset);
	}
}

__global__ void gIndexJoin(int *R, int *S, int g_locations[], int sLen, int g_ResNums[])
{
	int s_cur = blockIdx.x * blockDim.x + threadIdx.x;

	if(s_cur < sLen) 
	{
		int count = 1;
		int r_cur = g_locations[s_cur];
		int s_key;
		if(r_cur >= 0) /*&& r_cur < rLen) Tal vez la segunda parte no sea necesaria*/
		{
			s_key = S[s_cur];
			r_cur++;
			while(s_key == R[r_cur]) 
			{
				count++;
				r_cur++;
			}
			g_ResNums[s_cur] = count;
		}
		
	}
}

__global__ void gIndexMultiJoin(int *R, int *S, int g_locations[], int sLen, int g_ResNums[], int *p1, int *p2, int of1, int of2, int *mloc, int *sloc, int *muljoin, int wj)
{
	extern __shared__ int shared[];
	int s_cur = blockIdx.x * blockDim.x + threadIdx.x;
	int posr, poss, x, y, ini;

	if(threadIdx.x < wj)
		shared[threadIdx.x] = muljoin[threadIdx.x];
	__syncthreads();

	if(s_cur < sLen) 
	{
		int count = 1;
		int r_cur = g_locations[s_cur];
		int s_key;
		if(r_cur >= 0) /*&& r_cur < rLen) Tal vez la segunda parte no sea necesaria*/
		{
			s_key = S[s_cur];				
			r_cur++;
			while(s_key == R[r_cur]) 
			{
				count++;
				r_cur++;
			}
			if(sloc == NULL)
				poss = s_cur * of2;
			else
				poss = sloc[s_cur] * of2;
			ini = r_cur - count;	
			for(y = ini; y < r_cur; y++)
			{
				posr = mloc[y] * of1;
				for(x = 0; x < wj; x += 2)
				{
					if(p1[posr + shared[x]] != p2[poss + shared[x+1]])
					{
						count--;
						break;
					}
				}
			}
			if(count > 0)
				g_ResNums[s_cur] = count;
		}
		
	}
}

__global__ void multiJoinWithWrite(int g_locations[], int sLen, int g_PrefixSums[], int g_joinResultBuffers[], int *p1, int *p2, int of1, int of2, int *rule, int halfrul, int lenrul, int *mloc, int *sloc, int wj)
{
	extern __shared__ int shared[];
	int *extjoins = &shared[lenrul];
	int s_cur = blockIdx.x * blockDim.x + threadIdx.x;

	if(threadIdx.x < (lenrul + wj))
		shared[threadIdx.x] = rule[threadIdx.x];
	__syncthreads();

	if(s_cur < sLen)
	{
		int r_cur = g_locations[s_cur];
		if(r_cur >= 0)
		{
			int x, y, pos, posr, poss;
			int num1 = g_PrefixSums[s_cur];
			int num2 = g_PrefixSums[s_cur+1];
			
			int tmp1, tmp2;

			if(sloc == NULL)	
				poss = s_cur * of2;
			else
				poss = sloc[s_cur] * of2;
			for(x = num1; x < num2; x++, r_cur++)
			{
				pos = mloc[r_cur] * of1;
				for(y = 0; y < wj; y += 2)
				{
					tmp1 = p1[pos + extjoins[y]];
					tmp2 = p2[poss + extjoins[y+1]];
					if(tmp1 != tmp2)
						break;
				}
				if(y < wj)
				{
					x--;
					continue;
				}
				posr = x * lenrul;
				for(y = 0; y < halfrul; y++)
					g_joinResultBuffers[posr + y] = p1[pos + shared[y]];
				for(; y < lenrul; y++)
					g_joinResultBuffers[posr + y] = p2[poss + shared[y]];
			}
		}
	}
}

__global__ void multiJoinWithWrite2(int g_locations[], int sLen, int g_PrefixSums[], int g_joinResultBuffers[], int *p1, int *p2, int of1, int of2, int *rule, int cols, int *mloc, int *sloc, int wj)
{
	extern __shared__ int shared[];
	int *extjoins = &shared[cols];
	int s_cur = blockIdx.x * blockDim.x + threadIdx.x;

	if(threadIdx.x < (cols + wj))
		shared[threadIdx.x] = rule[threadIdx.x];
	__syncthreads();

	if(s_cur < sLen)
	{
		int r_cur = g_locations[s_cur];
		if(r_cur >= 0)
		{
			int x, y, pos, pos2, posr, cond;
			int num1 = g_PrefixSums[s_cur];
			int num2 = g_PrefixSums[s_cur+1];
			if(sloc == NULL)
				pos2 = s_cur * of2 - 1;
			else
				pos2 = sloc[s_cur] * of2 - 1;
			for(x = num1; x < num2; x++, r_cur++)
			{
				pos = mloc[r_cur] * of1 - 1;
				for(y = 0; y < wj; y += 2)
				{
					if(p1[pos + extjoins[y] + 1] != p2[pos2 + extjoins[y+1] + 1])
						break;
				}
				if(y < wj)
				{
					x--;
					continue;
				}
				posr = x * cols;
				for(y = 0; y < cols; y++)
				{
					cond = shared[y];
					if(cond > 0)
						g_joinResultBuffers[posr + y] = p1[pos + cond];
					else
						g_joinResultBuffers[posr + y] = p2[pos2 - cond];
				}
			}
		}
	}
}

__global__ void gJoinWithWrite(int g_locations[], int sLen, int g_PrefixSums[], int g_joinResultBuffers[], int *p1, int *p2, int of1, int of2, int *rule, int halfrul, int lenrul, int *mloc, int *sloc)
{
	extern __shared__ int shared[];
	int s_cur = blockIdx.x * blockDim.x + threadIdx.x;

	if(threadIdx.x < lenrul)
		shared[threadIdx.x] = rule[threadIdx.x];
	__syncthreads();

	if(s_cur < sLen)
	{
		int r_cur = g_locations[s_cur];
		if(r_cur >= 0)
		{
			int x, y, pos, posr, poss;
			int num1 = g_PrefixSums[s_cur];
			int num2 = g_PrefixSums[s_cur+1];
			if(sloc == NULL)	
				poss = s_cur * of2;
			else
				poss = sloc[s_cur] * of2;
			for(x = num1; x < num2; x++, r_cur++)
			{
				pos = mloc[r_cur] * of1;
				posr = x * lenrul;
				for(y = 0; y < halfrul; y++)
					g_joinResultBuffers[posr + y] = p1[pos + shared[y]];
				for(; y < lenrul; y++)
					g_joinResultBuffers[posr + y] = p2[poss + shared[y]];
			}
		}
	}
}

__global__ void gJoinWithWrite2(int g_locations[], int sLen, int g_PrefixSums[], int g_joinResultBuffers[], int *p1, int *p2, int of1, int of2, int *rule, int cols, int *mloc, int *sloc)
{
	extern __shared__ int shared[];
	int s_cur = blockIdx.x * blockDim.x + threadIdx.x;

	if(threadIdx.x < cols)
		shared[threadIdx.x] = rule[threadIdx.x];
	__syncthreads();

	if(s_cur < sLen)
	{
		int r_cur = g_locations[s_cur];
		if(r_cur >= 0)
		{
			int x, y, pos, pos2, posr, cond;
			int num1 = g_PrefixSums[s_cur];
			int num2 = g_PrefixSums[s_cur+1];
			if(sloc == NULL)
				pos2 = s_cur * of2 - 1;
			else
				pos2 = sloc[s_cur] * of2 - 1;
			for(x = num1; x < num2; x++, r_cur++)
			{
				pos = mloc[r_cur] * of1 - 1;
				posr = x * cols;
				for(y = 0; y < cols; y++)
				{
					cond = shared[y];
					if(cond > 0)
						g_joinResultBuffers[posr + y] = p1[pos + cond];
					else
						g_joinResultBuffers[posr + y] = p2[pos2 - cond];
				}
			}
		}
	}
}

int compare2 (const void * a, const void * b)
{
  return ( ((int2*)a)->y - ((int2*)b)->y );
}

/*void generateSort(Record *R, int maxmax, int rLen, int seed)
{
	int i=0;
	const int offset=(1<<15)-1;
	srand(seed);
	for(i=0;i<rLen;i++)
	{
		R[i].y=((((rand()& offset)<<15)+(rand()&1))+(rand()<<1)+(rand()&1))%maxmax;
		
	}
	qsort(R,rLen,sizeof(Record),compare);
	for(i=0;i<rLen;i++)
	R[i].x=i;

}

void generateRand(Record *R, int maxmax, int rLen, int seed)
{
	int i=0;
	const int offset=(1<<15)-1;
	srand(seed);
	for(i=0;i<rLen;i++)
	{
		R[i].y=((((rand()& offset)<<15)+(rand()&1))+(rand()<<1)+(rand()&1))%maxmax;
		//R[i].x=i+1;
		R[i].x=i;
	}
}*/

__global__ void llenar(int *p, int *R, int len, int of, int wj, int *pos, int *ids)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int cond;
	if(id < len)
	{
		cond = pos[id+1];
		if(pos[id] != cond && cond > 0)
		{
			R[cond-1] = p[id * of + wj];
			ids[cond-1] = id;
		}
	}
}

__global__ void llenar2(int *p, int *R, int len, int of, int wj, int *pos)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int cond;
	if(id < len)
	{
		cond = pos[id+1];
		if(pos[id] != cond && cond > 0)
			R[cond-1] = p[id * of + wj];
	}
}

__global__ void llenarnosel(int *p, int *R, int len, int of, int wj)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < len)
		R[id] = p[id * of + wj];
}

/*__global__ smalljoinc(int *p1, int *p2, int rLen, int sLen, int2 wj, int *r)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < rlen)
	{
		int comp = p1[id * of1 + wj.x];
		int x, cnt = 0;
		for(x = 0; x < sLen; x++)
		{
			if(comp == p2[x * of2 ])
				cnt++;
		}
		r[id] = cnt;
	}
}

__global__ smalljoinw(int *p1, int *p2, int wj, int *r)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < rlen)
	{
		int comp = p1[id];
		int x;
		for(x = 0; x < sLen; x++)
		{
			if(comp == p2[x])
				cnt++;
		}
		r[id] = cnt;
	}
}*/

int buscarunion(int *tmprule, int tmplen, int *rule, int pos2, int tam2, int *joins)
{
	int x, y;
	int cont = 0;
	for(x = 0; x < tmplen; x++)
	{
		for(y = pos2; y < (pos2 + tam2); y++)
		{
			if(tmprule[x] == rule[y])
			{
				joins[cont] = x;
				cont++;
				joins[cont] = y - pos2;
				cont++;
			}
		}
	}
	return cont;
}

int not_in(int *rule, int len, int bus)
{
	int x;
	for(x = 0; x < len; x++)
	{
		if(rule[x] == bus)
			return 0;
	}
	return 1;
}

int posiciones(int ini, int of1, int *firstpart, int *hpos, int *hcons, int *rule, int *temprule, int *tmprulpos, int *lenrul, int pos2, int of2, int posf)
{
	int cont = 0, x, y;
	for(y = ini; y < of1; y++)
	{
		if(firstpart[y] < 0)
		{
			hpos[cont] = y - ini;
			hcons[cont] = -firstpart[y];
			cont++;
		}
		else
		{
			x = 1;
			while(rule[x] != 0)
			{
				if(firstpart[y] == rule[x])
				{
					if(not_in(temprule, *lenrul, firstpart[y]))
					{
						temprule[*lenrul] = firstpart[y];
						tmprulpos[*lenrul] = y - ini;
						*lenrul = *lenrul + 1;
					}
					break;
				}
				x++;
			}
			if(rule[x] != 0)
				continue;
			for(x = (pos2 + of2 + 1); x < posf; x++)
			{
				if(rule[x] == 0)
				{
					x++;
					continue;
				}
				if(firstpart[y] == rule[x])
				{
					if(not_in(temprule, *lenrul, firstpart[y]))
					{
						temprule[*lenrul] = firstpart[y];
						tmprulpos[*lenrul] = y - ini;
						*lenrul = *lenrul + 1;
					}
					break;
				}
			}
		}
	}	
	return cont;
}

void join_final(int cols, int of1, int of2, int *rule, int *firstpart, int *secondpart, int *tmprulpos)
{
	int x, y;
	for(y = 0; y < cols; y++)
	{
		for(x = 0; x < of1; x++)
		{
			if(rule[y] == firstpart[x])
			{
				tmprulpos[y] = x + 1;
				break;
			}
		}
		if(x != of1)
			continue;
		for(x = 0; x < of2; x++)
		{
			if(rule[y] == secondpart[x])
			{
				tmprulpos[y] = -x - 1;
				break;
			}
		}
	}
}

int select_pos(int of1, int *firstpart, int *hpos, int *hcons)
{
	int cont = 0, y;
	for(y = 0; y < of1; y++)
	{
		if(firstpart[y] < 0)
		{
			hpos[cont] = y;
			hcons[cont] = -firstpart[y];
			cont++;
		}
	}
	return cont;
}

int checkquery(int *hpos, int *hcons, int cont, int *pred, int of1, int *rule, int *query, int cols)
{
	int x, y;
	if(rule[0] != query[0])
		return cont;
	for(x = 1; x <= cols; x++)
	{
		if(query[x] < 0)
		{
			for(y = 0; y < of1; y++)
			{
				if(pred[y] == rule[x])
				{
					hpos[cont] = y;
					hcons[cont] = -query[x];
					cont++;
				}
			}
		}
	}
	return cont;
}

int maximo(int count, ...)
{
	va_list ap;
    	int j, temp, mx = 0;
    	va_start(ap, count);

	for(j = 0; j < count; j++)
	{
		temp = va_arg(ap, int);
		if(temp > mx)
			mx = temp;
	}

    	va_end(ap);
    	return mx;
}

template <typename KeyVector, typename PermutationVector, typename TempVector>
void update_permutation(KeyVector& keys, PermutationVector& permutation, TempVector& temporary, int rows)
{
    // permute the keys with the current reordering
	thrust::gather(permutation, permutation + rows, keys, temporary);

    // stable_sort the permuted keys and update the permutation
	thrust::stable_sort_by_key(temporary, temporary + rows, permutation);
}


template <typename KeyVector, typename PermutationVector, typename TempVector>
void apply_permutation(KeyVector& keys, PermutationVector& permutation, TempVector& temporary, int rows)
{
    // permute the keys
    thrust::gather(permutation, permutation + rows, temporary, keys);
}

int join(int *p1, int *p2, int rLen, int sLen, int of1, int of2, list<rulenode>::iterator rule, int pos, int bothops, int **ret)
{
	int pos2 = pos + 1;
	int *sel1, nsel1;
	int *sel2 = rule->select[pos2];
	int nsel2 = rule->numsel[pos2];
	int *proj = rule->project[pos];
	int2 projp = rule->projpos[pos];
	int *sjoin1, nsj1;
	int *sjoin2 = rule->selfjoin[pos2];
	int nsj2 = rule->numselfj[pos2];
	int *wherej = rule->wherejoin[pos];
	int numj = rule->numjoin[pos];
	int flag;
#if TIMER
	cuda_stats.joins++;
#endif

	int porLiberar = rLen * of1 * sizeof(int);
	int size, sizet, sizet2;
	if(bothops)
	{
		sel1 = rule->select[pos];
		nsel1 = rule->numsel[pos];
		sjoin1 = rule->selfjoin[pos];
		nsj1 = rule->numselfj[pos];
		sizet = maximo(7, of1, of2, nsel1, nsel2, projp.y + numj - 2, nsj1, nsj2) * sizeof(int);
	}
	else
		sizet = maximo(6, of1, of2, nsel2, projp.y + numj - 2, nsj2, numj) * sizeof(int);
	
	int *dcons, *temp;
	int *d_R, *d_S;
	int blockllen, numthreads;
	
	//int por_liberar = rLen * sizeof(int);
	int extraspace = TREE_NODE_SIZE - rLen % TREE_NODE_SIZE;
	int m32rLen = rLen + extraspace;
	if(m32rLen > sLen)
		sizet2 = (m32rLen + 1) * sizeof(int);
	else
		sizet2 = (sLen + 1) * sizeof(int);

	/*hcons = (int *)malloc(sizet);
	hpos = (int *)malloc(sizet);
	int dconsize = sizet * 2;*/

	reservar(&dcons, sizet);
#ifdef DEBUG_MEM
	cerr << "+ " << dcons << " dcons tree  " << sizet << endl;
#endif
	reservar(&temp, sizet2);
#ifdef DEBUG_MEM
	cerr << "+ " << temp << " temp tree " << sizet2 << endl;
#endif
	thrust::device_ptr<int> res = thrust::device_pointer_cast(temp);

	numthreads = 1024;
	blockllen = sLen / numthreads + 1;
	int memSizeS, newLen;
	int *posR = NULL, *posS = NULL;

	#ifdef TIMER
	//cout << "INICIO" << endl;
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	#endif

	//cout << "sLen y rLen = " << sLen << " " << rLen << endl;

	if(nsel2 > 0)
	{
		size = nsel2 * sizeof(int);
		newLen = sLen + 1;
		hipMemsetAsync(temp, 0, newLen * sizeof(int));
		hipMemcpy(dcons, sel2, size, hipMemcpyHostToDevice);
		marcar<<<blockllen, numthreads, size>>>(p2, sLen, of2, dcons, nsel2, temp + 1);

		/*int y;
		int *htemp = (int *)malloc(newLen * sizeof(int));
		cout << "temp =" << endl;
		hipMemcpy(htemp, temp, newLen * sizeof(int), hipMemcpyDeviceToHost);
		for(y = 0; y < newLen; y++)
			cout << htemp[y] << " ";
		cout << endl;
		free(htemp);*/

		if(nsj2 > 0)
		{
			size = nsj2 * sizeof(int);
			hipMemcpy(dcons, sjoin2, size, hipMemcpyHostToDevice);
			samejoin<<<blockllen, numthreads, size>>>(p2, sLen, of2, dcons, nsj2, temp + 1);
		}

		/*htemp = (int *)malloc(newLen * sizeof(int));
		cout << "temp =" << endl;
		hipMemcpy(htemp, temp, newLen * sizeof(int), hipMemcpyDeviceToHost);
		for(y = 0; y < newLen; y++)
			cout << res[y] << " ";
		cout << endl;
		free(htemp);*/

		/*cout << "Despues de marcar" << endl;
		cout << "newLen = " << newLen << endl;*/

		flag = 0;
		while(flag != 1)
		{
			try
			{
				thrust::inclusive_scan(res + 1, res + newLen, res + 1);		
				flag = 1;
			}
			catch(std::bad_alloc &e)
			{
				limpiar("inclusive scan in join", 0);
			}				
		}
		//thrust::inclusive_scan(res + 1, res + newLen, res + 1);	
		newLen = res[sLen];
	
		if(newLen == 0)
			return 0;

		memSizeS = newLen * sizeof(int);
		reservar(&d_S, memSizeS);
#if DEBUG_MEM
		cerr << "+ " << d_S << " d_S  " << memSizeS << endl;
#endif
		reservar(&posS, memSizeS);
#if DEBUG_MEM
		cerr << "+ " << posS << " posS  " << memSizeS << endl;
#endif
		llenar<<<blockllen, numthreads>>>(p2, d_S, sLen, of2, wherej[1], temp, posS);
		sLen = newLen;
	}
	else
	{
		if(nsj2 > 0)
		{
			size = nsj2 * sizeof(int);
			newLen = sLen + 1;
			hipMemsetAsync(temp, 0, newLen * sizeof(int));
			hipMemcpy(dcons, sjoin2, size, hipMemcpyHostToDevice);
			samejoin2<<<blockllen, numthreads, size>>>(p2, sLen, of2, dcons, nsj2, temp + 1);

			thrust::inclusive_scan(res + 1, res + newLen, res + 1);
			newLen = res[sLen];
			if(newLen == 0)
			  return 0;

			memSizeS = newLen * sizeof(int);
			reservar(&d_S, memSizeS);
#ifdef DEBUG_MEM
			cerr << "+ " << d_S << " d_S m " << memSizeS << endl;
#endif
			reservar(&posS, memSizeS);
#ifdef DEBUG_MEM
			cerr << "+ " << posS << " posS m " << memSizeS << endl;
#endif
			llenar<<<blockllen, numthreads>>>(p2, d_S, sLen, of2, wherej[1], temp, posS);
			sLen = newLen;
		}
		else
		{
			memSizeS = sLen * sizeof(int);
			reservar(&d_S, memSizeS);
#ifdef DEBUG_MEM
			cerr << "+ " << d_S << " d_S n " << memSizeS << endl;
#endif
			llenarnosel<<<blockllen, numthreads>>>(p2, d_S, sLen, of2, wherej[1]);
		}
	}

	#ifdef TIMER
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	//cout << "Select1 = " << time << endl;
	cuda_stats.select1_time += time;

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	#endif

	blockllen = rLen / numthreads + 1;
	int sizem32, sizextra;
	if(bothops)
	{
		if(nsel1 > 0)
		{
			size = nsel1 * sizeof(int);
			newLen = rLen + 1;
			hipMemsetAsync(temp, 0, newLen * sizeof(int));
			hipMemcpy(dcons, sel1, size, hipMemcpyHostToDevice);
			marcar<<<blockllen, numthreads, size>>>(p1, rLen, of1, dcons, nsel1, temp + 1);

			if(nsj1 > 0)
			{
				size = nsj1 * sizeof(int);
				hipMemcpy(dcons, sjoin1, size, hipMemcpyHostToDevice);
				samejoin<<<blockllen, numthreads, size>>>(p1, rLen, of1, dcons, nsj1, temp + 1);
			}

			thrust::inclusive_scan(res + 1, res + newLen, res + 1);
			newLen = res[rLen];
			if(newLen == 0)
				return 0;		

			extraspace = TREE_NODE_SIZE - newLen % TREE_NODE_SIZE;
			sizextra = extraspace * sizeof(int);
			m32rLen = newLen + extraspace;
			sizem32 = m32rLen * sizeof(int);
			reservar(&d_R, sizem32);
#ifdef DEBUG_MEM
			cerr << "+ " << d_R << " d_R m " << sizem32 << endl;
#endif
			reservar(&posR, sizem32);
#ifdef DEBUG_MEM
			cerr << "+ " << posR << " posR m " << sizem32 << endl;
#endif
			hipMemsetAsync(d_R + newLen, 0x7f, sizextra);
			hipMemsetAsync(posR + newLen, 0x7f, sizextra);
			llenar<<<blockllen, numthreads>>>(p1, d_R, rLen, of1, wherej[0], temp, posR);
			rLen = newLen;
		}
		else
		{
			if(nsj1 > 0)
			{
				size = nsj1 * sizeof(int);
				newLen = rLen + 1;
				hipMemsetAsync(temp, 0, newLen * sizeof(int));
				hipMemcpy(dcons, sjoin1, size, hipMemcpyHostToDevice);
				samejoin2<<<blockllen, numthreads, size>>>(p1, rLen, of1, dcons, nsj1, temp + 1);

				thrust::inclusive_scan(res + 1, res + newLen, res + 1);
				newLen = res[rLen];
				if(newLen == 0)
					return 0;

				extraspace = TREE_NODE_SIZE - newLen % TREE_NODE_SIZE;
				sizextra = extraspace * sizeof(int);
				m32rLen = newLen + extraspace;
				sizem32 = m32rLen * sizeof(int);
				reservar(&d_R, sizem32);
#ifdef DEBUG_MEM
				cerr << "+ " << d_R << " d_R n " << sizem32 << endl;
#endif
				reservar(&posR, sizem32);
#ifdef DEBUG_MEM
				cerr << "+ " << posR << " posR n " << sizem32 << endl;
#endif
				hipMemsetAsync(d_R + newLen, 0x7f, sizextra);
				hipMemsetAsync(posR + newLen, 0x7f, sizextra);
				llenar<<<blockllen, numthreads>>>(p1, d_R, rLen, of1, wherej[0], temp, posR);
				rLen = newLen;
			}
			else
			{
				sizem32 = m32rLen * sizeof(int);
				reservar(&d_R, sizem32);
#ifdef DEBUG_MEM
				cerr << "+ " << d_R << " d_R sizem32 " << sizem32 << endl;
#endif
				hipMemsetAsync(d_R + rLen, 0x7f, extraspace * sizeof(int));
				llenarnosel<<<blockllen, numthreads>>>(p1, d_R, rLen, of1, wherej[0]);
			}
			
		}
	}
	else
	{
		sizem32 = m32rLen * sizeof(int);
		reservar(&d_R, sizem32);
#ifdef DEBUG_MEM
		cerr << "+ " << d_R << " d_R sz " << sizem32 << endl;
#endif
		hipMemsetAsync(d_R + rLen, 0x7f, extraspace * sizeof(int));
		llenarnosel<<<blockllen, numthreads>>>(p1, d_R, rLen, of1, wherej[0]);
	}

	#ifdef TIMER
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	//cout << "Select2 = " << time << endl;
	cuda_stats.select2_time += time;
	#endif
	
	/*free(hcons);
	free(hpos);

	h_R = (int *)malloc(sizem32);
	hipMemcpy(h_R, d_S, memSizeS, hipMemcpyDeviceToHost);
	cout << "H_S " << "cont " << cont << " sLen " << sLen << endl;
	for(x = 0; x < sLen; x++)
		cout << h_R[x] << endl;
	free(h_R);*/

	#ifdef TIMER
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	#endif

	thrust::device_ptr<Record> dvp1 = thrust::device_pointer_cast(d_R);
	thrust::device_ptr<Record> permutation;
	if(posR == NULL)
	{
		reservar(&posR, sizem32);
#ifdef DEBUG_MEM
		cerr << "+ " << posR << " posR m32 " << sizem32 << endl;
#endif
		permutation = thrust::device_pointer_cast(posR);
		thrust::sequence(permutation, permutation + m32rLen);
	}
	else
		permutation = thrust::device_pointer_cast(posR);

	flag = 0;
	while(flag != 1)
	{
		try
		{
			thrust::stable_sort_by_key(dvp1, dvp1 + m32rLen, permutation);
			flag = 1;
		}
		catch(std::bad_alloc &e)
		{
			limpiar("inclusive scan in join", 0);
		}
	}

	#ifdef TIMER
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	//cout << "Sort = " << time << endl;
	cuda_stats.sort_time += time;
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	#endif

	IDataNode* d_data;
	IDirectoryNode* d_dir;
	unsigned int nDataNodes;

	nDataNodes = uintCeilingDiv(rLen, TREE_NODE_SIZE);
	d_data=(IDataNode *)d_R;

	unsigned int lvlDir = uintCeilingLog(TREE_FANOUT, nDataNodes);
	unsigned int nDirNodes = uintCeilingDiv(nDataNodes - 1, TREE_NODE_SIZE);
	unsigned int tree_size = nDirNodes + nDataNodes;
	unsigned int bottom_start = (uintPower(TREE_FANOUT, lvlDir) - 1) / TREE_NODE_SIZE;
	d_dir = (IDirectoryNode *)temp;

	unsigned int nNodesPerBlock = uintCeilingDiv(nDirNodes, BLCK_PER_GRID_create);

	dim3 Dbc(THRD_PER_BLCK_create, 1, 1);
	dim3 Dgc(BLCK_PER_GRID_create, 1, 1);

	gCreateIndex <<<Dgc, Dbc>>> (d_data, d_dir, nDirNodes, tree_size, bottom_start, nNodesPerBlock);

	/*int y;
	IDirectoryNode *h_dir = (IDirectoryNode*)malloc(sizeof(IDirectoryNode) * nDirNodes);
	hipMemcpy(h_dir, d_dir, sizeof(IDirectoryNode) * nDirNodes, hipMemcpyDeviceToHost);
	for(x = 0; x < nDirNodes; x++)
	{
		for(y = 0; y < TREE_NODE_SIZE; y++)
			printf("%d ", h_dir[x].keys[y]);
		printf("\n");
	}
	free(h_dir);*/

	int *d_locations;
	reservar(&d_locations, memSizeS);
#ifdef DEBUG_MEM
	cerr << "+ " << d_locations << " d_locs n " << memSizeS << endl;
#endif

	dim3 Dbs(THRD_PER_BLCK_search, 1, 1);
	dim3 Dgs(BLCK_PER_GRID_search, 1, 1);

	unsigned int nSearchKeys = sLen;
	unsigned int nKeysPerThread = uintCeilingDiv(nSearchKeys, THRD_PER_GRID_search);

	gSearchTree <<<Dgs, Dbs>>> (d_data, nDataNodes, d_dir, nDirNodes, lvlDir, d_S, d_locations, nSearchKeys, nKeysPerThread, tree_size, bottom_start);
	hipMemsetAsync(temp, 0, memSizeS);

	blockllen = sLen / numthreads + 1;
	int muljoin = 0, muljoinsize = 0;
	if(numj > 2)
	{
		muljoin = numj - 2;
		muljoinsize = muljoin * sizeof(int);
		hipMemcpy(dcons, wherej + 2, muljoinsize, hipMemcpyHostToDevice);
		gIndexMultiJoin<<<blockllen, numthreads, muljoinsize>>> (d_R, d_S, d_locations, sLen, temp, p1, p2, of1, of2, posR, posS, dcons, muljoin);
	}
	else
		gIndexJoin<<<blockllen, numthreads>>> (d_R, d_S, d_locations, sLen, temp);
	liberar(d_R, sizem32);
	liberar(d_S, memSizeS);

	int sum = res[sLen-1];
	thrust::exclusive_scan(res, res + sLen, res);
	sum += res[sLen-1];
	if(sum == 0)
		return 0;	
	res[sLen] = sum;

	int *d_Rout;
	int resSize, sizepro;
	if(pos == (rule->num_rows - 3) && rule->num_bpreds.x == 0)
	{
		sizepro = rule->num_columns * sizeof(int);
		hipMemcpy(dcons, proj, sizepro, hipMemcpyHostToDevice);
		resSize = sum * sizepro;
		reservar(&d_Rout, resSize);
#ifdef DEBUG_MEM
		cerr << "+ " << d_Rout << " d_Rout n " << resSize << endl;
#endif
		if(numj > 2)
		{
			hipMemcpy(dcons + rule->num_columns, wherej + 2, muljoinsize, hipMemcpyHostToDevice);
			multiJoinWithWrite2<<<blockllen, numthreads, sizepro + muljoinsize>>> (d_locations, sLen, temp, d_Rout, p1, p2, of1, of2, dcons, rule->num_columns, posR, posS, muljoin);
		}
		else
			gJoinWithWrite2<<<blockllen, numthreads, sizepro>>> (d_locations, sLen, temp, d_Rout, p1, p2, of1, of2, dcons, rule->num_columns, posR, posS);
	}
	else
	{
		sizepro = projp.y * sizeof(int);
		hipMemcpy(dcons, proj, sizepro, hipMemcpyHostToDevice);
		resSize = sum * sizepro;
		reservar(&d_Rout, resSize);
#ifdef DEBUG_MEM
		cerr << "+ " << d_Rout << " d_Rout 2 " << resSize << endl;
#endif
		if(numj > 2)
		{
			hipMemcpy(dcons + projp.y, wherej + 2, muljoinsize, hipMemcpyHostToDevice);
			multiJoinWithWrite<<<blockllen, numthreads, sizepro + muljoinsize>>> (d_locations, sLen, temp, d_Rout, p1, p2, of1, of2, dcons, projp.x, projp.y, posR, posS, muljoin);
		}
		else
			gJoinWithWrite<<<blockllen, numthreads, sizepro>>> (d_locations, sLen, temp, d_Rout, p1, p2, of1, of2, dcons, projp.x, projp.y, posR, posS);
	}

	liberar(dcons, sizet);
	liberar(d_locations, memSizeS);
	liberar(temp, sizet2);
	liberar(posR, sizem32);
	if(posS != NULL)
		liberar(posS, memSizeS);
	
	/*if(posS != NULL)
		liberar(posS, memSizeS);
	liberar(dtmprulpos, sizerul);
	if(*ret != NULL)
		liberar(*ret, por_liberar);
	free(tmprulpos);
	if(final_cond != posf)
	{
		free(*newrule);
		*newrule = (int *)malloc(sizerul);
		memcpy(*newrule, temprule, sizerul);
		*newrullen = lenrul;
	}*/
	
	if(*ret != NULL)
		liberar(*ret, porLiberar);
	*ret = d_Rout;

	#ifdef TIMER
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	//cout << "Join = " << time << endl;
	//cout << "FIN" << endl;
	cuda_stats.join_time += time;
	#endif

	return sum;
}
