#include <thrust/device_vector.h>
#include <thrust/unique.h>
#include <thrust/distance.h>
#include <iostream>

typedef struct n2
{
	int v[2];
}s2;

typedef struct n3
{
	int v[3];
}s3;

typedef struct n4
{
	int v[4];
}s4;

typedef struct n5
{
	int v[5];
}s5;

typedef struct n6
{
	int v[6];
}s6;

typedef struct n7
{
	int v[7];
}s7;

struct p2
{
	__host__ __device__
    	bool operator()(const s2 &r1, const s2 &r2)
    	{
     		int x;
		for(x = 0; x < 2; x++)
		{
			if(r1.v[x] != r2.v[x])
				return false;
		}
		return true;
    	}
};

struct o2
{
	__host__ __device__
    	bool operator()(const s2 &r1, const s2 &r2)
    	{
     		int x;
		for(x = 0; x < 2; x++)
		{
			if(r1.v[x] > r2.v[x])
				return true;
			if(r1.v[x] < r2.v[x])
				return false;
		}
		return false;
    	}
};

struct p3
{
	__host__ __device__
    	bool operator()(const s3 &r1, const s3 &r2)
    	{
     		int x;
		for(x = 0; x < 3; x++)
		{
			if(r1.v[x] != r2.v[x])
				return false;
		}
		return true;
    	}
};

struct o3
{
	__host__ __device__
    	bool operator()(const s3 &r1, const s3 &r2)
    	{
     		int x;
		for(x = 0; x < 3; x++)
		{
			if(r1.v[x] > r2.v[x])
				return true;
			if(r1.v[x] < r2.v[x])
				return false;
		}
		return false;
    	}
};

struct p4
{
	__host__ __device__
    	bool operator()(const s4 &r1, const s4 &r2)
    	{
     		int x;
		for(x = 0; x < 4; x++)
		{
			if(r1.v[x] != r2.v[x])
				return false;
		}
		return true;
    	}
};

struct o4
{
	__host__ __device__
    	bool operator()(const s4 &r1, const s4 &r2)
    	{
     		int x;
		for(x = 0; x < 4; x++)
		{
			if(r1.v[x] > r2.v[x])
				return true;
			if(r1.v[x] < r2.v[x])
				return false;
		}
		return false;
    	}
};

struct p5
{
	__host__ __device__
    	bool operator()(const s5 &r1, const s5 &r2)
    	{
     		int x;
		for(x = 0; x < 5; x++)
		{
			if(r1.v[x] != r2.v[x])
				return false;
		}
		return true;
    	}
};

struct o5
{
	__host__ __device__
    	bool operator()(const s5 &r1, const s5 &r2)
    	{
     		int x;
		for(x = 0; x < 5; x++)
		{
			if(r1.v[x] > r2.v[x])
				return true;
			if(r1.v[x] < r2.v[x])
				return false;
		}
		return false;
    	}
};

struct p6
{
	__host__ __device__
    	bool operator()(const s6 &r1, const s6 &r2)
    	{
     		int x;
		for(x = 0; x < 6; x++)
		{
			if(r1.v[x] != r2.v[x])
				return false;
		}
		return true;
    	}
};

struct o6
{
	__host__ __device__
    	bool operator()(const s6 &r1, const s6 &r2)
    	{
     		int x;
		for(x = 0; x < 6; x++)
		{
			if(r1.v[x] > r2.v[x])
				return true;
			if(r1.v[x] < r2.v[x])
				return false;
		}
		return false;
    	}
};

struct p7
{
	__host__ __device__
    	bool operator()(const s7 &r1, const s7 &r2)
    	{
     		int x;
		for(x = 0; x < 7; x++)
		{
			if(r1.v[x] != r2.v[x])
				return false;
		}
		return true;
    	}
};

struct o7
{
	__host__ __device__
    	bool operator()(const s7 &r1, const s7 &r2)
    	{
     		int x;
		for(x = 0; x < 7; x++)
		{
			if(r1.v[x] > r2.v[x])
				return true;
			if(r1.v[x] < r2.v[x])
				return false;
		}
		return false;
    	}
};

int unir(int *res, int rows, int tipo)
{
	int flag, nrows;

#if TIMER
	cuda_stats.unions++;
#endif
	switch(tipo)
	{
		case 1: 
		{
			thrust::device_ptr<int> pt, re;

			pt = thrust::device_pointer_cast(res);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt, pt + rows);
					re = thrust::unique(pt, pt + rows);
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}				
			}
			nrows = thrust::distance(pt, re);
			thrust::device_vector<int> iVec(pt, pt + rows);
			iVec.resize(nrows);
			iVec.shrink_to_fit();
			return nrows;	
		}			
		case 2: 
		{
			thrust::device_ptr<s2> pt2, re2;
			s2 *t2;
			t2 = (s2*)res;
			
			/*int *a, x, y;
			a = (int *)malloc(rows * 2 * sizeof(int));
			hipMemcpy(a, res, rows * 2 * sizeof(int), hipMemcpyDeviceToHost);
			cout << "INI" << endl;
			for(x = 0; x < rows; x++)
			{
				for(y = 0; y < 2; y++)
					cout << a[x * 2 + y] << " ";
				cout << endl;	
			}
			cout << "INI fin" << endl;
			free(a);*/

			pt2 = thrust::device_pointer_cast(t2);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt2, pt2 + rows, o2());
					re2 = thrust::unique(pt2, pt2 + rows, p2());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}				
			}
			nrows = thrust::distance(pt2, re2);
			thrust::device_vector<s2> iVec(pt2, pt2 + rows);
			iVec.resize(nrows);
			iVec.shrink_to_fit();

			/*tam = (int)(re2.get() - pt2.get());
			a = (int *)malloc(tam * 2 * sizeof(int));
			hipMemcpy(a, res, tam * 2 * sizeof(int), hipMemcpyDeviceToHost);
			cout << "FIN" << endl;
			for(x = 0; x < tam; x++)
			{
				for(y = 0; y < 2; y++)
					cout << a[x * 2 + y] << " ";
				cout << endl;	
			}
			cout << "FIN fin" << endl;
			free(a);
			cout << "antes = " << rows << " despues = " << thrust::distance(pt2, re2) << endl;*/

			return nrows;
		}
		case 3: 
		{
			thrust::device_ptr<s3> pt3, re3;
			s3 *t3;
			t3 = (s3*)res;
			pt3 = thrust::device_pointer_cast(t3);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt3, pt3 + rows, o3());
					re3 = thrust::unique(pt3, pt3 + rows, p3());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}				
			}
			nrows = thrust::distance(pt3, re3);
			thrust::device_vector<s3> iVec(pt3, pt3 + rows);
			iVec.resize(nrows);
			iVec.shrink_to_fit();
			return nrows;
		}
		case 4: 
		{
			thrust::device_ptr<s4> pt4, re4;
			s4 *t4;
			t4 = (s4*)res;
			pt4 = thrust::device_pointer_cast(t4);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt4, pt4 + rows, o4());
					re4 = thrust::unique(pt4, pt4 + rows, p4());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}				
			}
			nrows = thrust::distance(pt4, re4);
			thrust::device_vector<s4> iVec(pt4, pt4 + rows);
			iVec.resize(nrows);
			iVec.shrink_to_fit();
			return nrows;
		}
		case 5: 
		{
			thrust::device_ptr<s5> pt5, re5;
			s5 *t5;
			t5 = (s5*)res;
			pt5 = thrust::device_pointer_cast(t5);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt5, pt5 + rows, o5());
					re5 = thrust::unique(pt5, pt5 + rows, p5());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}				
			}
			nrows = thrust::distance(pt5, re5);
			thrust::device_vector<s5> iVec(pt5, pt5 + rows);
			iVec.resize(nrows);
			iVec.shrink_to_fit();
			return nrows;
		}
		case 6: 
		{
			thrust::device_ptr<s6> pt6, re6;
			s6 *t6;
			t6 = (s6*)res;
			pt6 = thrust::device_pointer_cast(t6);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt6, pt6 + rows, o6());
					re6 = thrust::unique(pt6, pt6 + rows, p6());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}				
			}
			nrows = thrust::distance(pt6, re6);
			thrust::device_vector<s6> iVec(pt6, pt6 + rows);
			iVec.resize(nrows);
			iVec.shrink_to_fit();
			return nrows;
		}
		case 7: 
		{
			thrust::device_ptr<s7> pt7, re7;
			s7 *t7;
			t7 = (s7*)res;
			pt7 = thrust::device_pointer_cast(t7);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt7, pt7 + rows, o7());
					re7 = thrust::unique(pt7, pt7 + rows, p7());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir", 0);
				}				
			}
			nrows = thrust::distance(pt7, re7);
			thrust::device_vector<s7> iVec(pt7, pt7 + rows);
			iVec.resize(nrows);
			iVec.shrink_to_fit();
			return nrows;
		}
	default:
	  cerr << "Union: " << tipo << " columns are too many." << endl;
	  exit(1);
	}
	return 0;
}
